/*
 Copyright (C) 2006-2007 M.A.L. Marques
               2018 Susi Lehtola

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"
#include "xc_device.h"
#include "hipblas.h"


#define is_mgga(id)   ((id) == XC_FAMILY_MGGA || (id) == XC_FAMILY_HYB_MGGA)
#define is_gga(id)    ((id) == XC_FAMILY_GGA  || (id) == XC_FAMILY_HYB_GGA || is_mgga(id))
#define is_lda(id)    ((id) == XC_FAMILY_LDA  || is_gga(is))
#define safe_free(pt) if(pt != NULL) free(pt)

hipblasHandle_t cublas_handle;
double *device_funcs_coef;

void
xc_mix_func_init_cublas()
{
    checkCublas(__FILE__,__LINE__,hipblasCreate(&cublas_handle));
    checkCublas(__FILE__,__LINE__,hipblasSetPointerMode(cublas_handle,HIPBLAS_POINTER_MODE_DEVICE));
    checkCuda(__FILE__,__LINE__,hipMalloc(&device_funcs_coef,XC_MAX_FUNC_TERMS*sizeof(double)));
}

void
xc_mix_func_end_cublas()
{
    checkCublas(__FILE__,__LINE__,hipblasDestroy(cublas_handle));
    checkCuda(__FILE__,__LINE__,hipFree(device_funcs_coef));
}

void
xc_mix_func_offload(const xc_func_type *func, int np,
            const double *rho, const double *sigma, const double *lapl, const double *tau,
            double *zk, MGGA_OUT_PARAMS_NO_EXC(double *),
            hipStream_t stream)
{
  const xc_func_type *aux;
  double *zk_;
  double *vrho_, *vsigma_, *vlapl_, *vtau_;
  double *v2rho2_, *v2rhosigma_, *v2rholapl_, *v2rhotau_;
  double *v2sigma2_, *v2sigmalapl_, *v2sigmatau_;
  double *v2lapl2_, *v2lapltau_;
  double *v2tau2_;
  double *v3rho3_, *v3rho2sigma_, *v3rho2lapl_, *v3rho2tau_;
  double *v3rhosigma2_, *v3rhosigmalapl_, *v3rhosigmatau_;
  double *v3rholapl2_, *v3rholapltau_;
  double *v3rhotau2_;
  double *v3sigma3_, *v3sigma2lapl_, *v3sigma2tau_;
  double *v3sigmalapl2_, *v3sigmalapltau_;
  double *v3sigmatau2_;
  double *v3lapl3_, *v3lapl2tau_;
  double *v3lapltau2_;
  double *v3tau3_;

  int ii;
  /* CUDA status */
  hipError_t stat;

  const xc_dimensions *dim = &(func->dim);

  /* prepare buffers that will hold the results from the individual functionals */
  /* The model is:
   * - we allocate the output buffers to be twice the regular size:
   *   - the first np entries will be set to the results of the total functional
   *   - the second np entries will be used as buffers for the functional terms.
   * - hence the second np entries are the ones passed into the functional as 
   *   output buffer.
   * - we need to use some cuBlas operation to add the terms as all the (double*)
   *   arguments are device pointers that cannot be accessed directly in this 
   *   function.
   * Doing things this way we can use linear combinations of functionals without
   * having to do strange things to the API.
   */
  zk_ = NULL;  
  vrho_ = vsigma_ = vlapl_ = vtau_ = NULL;
  v2rho2_ = v2rhosigma_ = v2rholapl_ = v2rhotau_ = NULL;
  v2sigma2_ =  v2sigmalapl_ = v2sigmatau_ = NULL;
  v2lapl2_ = v2lapltau_ = NULL;
  v2tau2_ = NULL;
  v3rho3_ = v3rho2sigma_ = v3rho2lapl_ = v3rho2tau_ = NULL;
  v3rhosigma2_ = v3rhosigmalapl_ = v3rhosigmatau_ = NULL;
  v3rholapl2_ = v3rholapltau_ = NULL;
  v3rhotau2_ = NULL;
  v3sigma3_ = v3sigma2lapl_ = v3sigma2tau_ = NULL;
  v3sigmalapl2_ = v3sigmalapltau_ = NULL;
  v3sigmatau2_ = NULL;
  v3lapl3_ = v3lapl2tau_ = NULL;
  v3lapltau2_ = NULL;
  v3tau3_ = NULL;

  if(zk != NULL)
    zk_ = zk + np*dim->zk;

  if(vrho != NULL){
    vrho_ = vrho + np*dim->vrho;
    if(is_gga(func->info->family)){
      vsigma_ = vsigma + np*dim->vsigma;
    }
    if(is_mgga(func->info->family)){
      /* At the moment we always allocate the derivatives involving
         the laplacian, as some parts of Libxc do not take into
         account the XC_FLAGS_NEEDS_LAPLACIAN flag.
         if(func->info->flags & XC_FLAGS_NEEDS_LAPLACIAN){ */
      vlapl_ = vlapl + np*dim->vlapl;
      /* } */
      vtau_  = vtau  + np*dim->vtau;
    }
  }

  if(v2rho2 != NULL){
    v2rho2_ = v2rho2 + np*dim->v2rho2;
    if(is_gga(func->info->family)){
      v2rhosigma_  = v2rhosigma + np*dim->v2rhosigma;
      v2sigma2_    = v2sigma2   + np*dim->v2sigma2;
    }
    if(is_mgga(func->info->family)){
      v2rholapl_   = v2rholapl   + np*dim->v2rholapl;
      v2rhotau_    = v2rhotau    + np*dim->v2rhotau;
      v2sigmalapl_ = v2sigmalapl + np*dim->v2sigmalapl;
      v2sigmatau_  = v2sigmatau  + np*dim->v2sigmatau;
      v2lapl2_     = v2lapl2     + np*dim->v2lapl2;
      v2lapltau_   = v2lapltau   + np*dim->v2lapltau;
      v2tau2_      = v2tau2      + np*dim->v2tau2;
    }
  }

  if(v3rho3 != NULL){
    v3rho3_      = v3rho3 + np*dim->v3rho3;
    if(is_gga(func->info->family)){
      v3rho2sigma_ = v3rho2sigma + np*dim->v3rho2sigma;
      v3rhosigma2_ = v3rhosigma2 + np*dim->v3rhosigma2;
      v3sigma3_    = v3sigma3    + np*dim->v3sigma3;
    }
    if(is_mgga(func->info->family)){
      v3rho2lapl_     = v3rho2lapl     + np*dim->v3rho2lapl;
      v3rho2tau_      = v3rho2tau      + np*dim->v3rho2tau;
      v3rhosigmalapl_ = v3rhosigmalapl + np*dim->v3rhosigmalapl;
      v3rhosigmatau_  = v3rhosigmatau  + np*dim->v3rhosigmatau;
      v3rholapl2_     = v3rholapl2     + np*dim->v3rholapl2;
      v3rholapltau_   = v3rholapltau   + np*dim->v3rholapltau;
      v3rhotau2_      = v3rhotau2      + np*dim->v3rhotau2;
      v3sigma2lapl_   = v3sigma2lapl   + np*dim->v3sigma2lapl;
      v3sigma2tau_    = v3sigma2tau    + np*dim->v3sigma2tau;
      v3sigmalapl2_   = v3sigmalapl2   + np*dim->v3sigmalapl2;
      v3sigmalapltau_ = v3sigmalapltau + np*dim->v3sigmalapltau;
      v3sigmatau2_    = v3sigmatau2    + np*dim->v3sigmatau2;
      v3lapl3_        = v3lapl3        + np*dim->v3lapl3;
      v3lapl2tau_     = v3lapl2tau     + np*dim->v3lapl2tau;
      v3lapltau2_     = v3lapltau2     + np*dim->v3lapltau2;
      v3tau3_         = v3tau3         + np*dim->v3tau3;
    }
  }
  
  /* we now add the different components */
  checkCuda(__FILE__,__LINE__,hipMemcpyAsync(device_funcs_coef,func->mix_coef,
                              func->n_func_aux*sizeof(double),hipMemcpyHostToDevice,stream));
  checkCublas(__FILE__,__LINE__,hipblasSetStream(cublas_handle, stream));
  for(ii=0; ii<func->n_func_aux; ii++){
    if(zk != NULL)
      checkCuda(__FILE__,__LINE__,hipMemsetAsync(zk_, 0, dim->zk*np*sizeof(double), stream));
  
    if(vrho != NULL){
      assert(vsigma != NULL);
  
      checkCuda(__FILE__,__LINE__,hipMemsetAsync(vrho_,   0, dim->vrho  *np*sizeof(double), stream));
      if(is_gga(func->info->family)){
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(vsigma_, 0, dim->vsigma*np*sizeof(double), stream));
      }
      if(is_mgga(func->info->family)){
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(vlapl_,  0, dim->vlapl*np*sizeof(double), stream));
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(vtau_,   0, dim->vtau*np*sizeof(double), stream));
      }
    }
  
    if(v2rho2 != NULL){
      assert(v2rhosigma!=NULL && v2sigma2!=NULL);
  
      checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2rho2_,        0, dim->v2rho2     *np*sizeof(double), stream));
      if(is_gga(func->info->family)){
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2rhosigma_,  0, dim->v2rhosigma *np*sizeof(double), stream));
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2sigma2_,    0, dim->v2sigma2   *np*sizeof(double), stream));
      }
      if(is_mgga(func->info->family)){
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2rholapl_,   0, dim->v2rholapl  *np*sizeof(double), stream));
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2rhotau_,    0, dim->v2rhotau   *np*sizeof(double), stream));
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2sigmalapl_, 0, dim->v2sigmalapl*np*sizeof(double), stream));
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2sigmatau_,  0, dim->v2sigmatau *np*sizeof(double), stream));
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2lapl2_,     0, dim->v2lapl2    *np*sizeof(double), stream));
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2lapltau_,   0, dim->v2lapltau  *np*sizeof(double), stream));
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2tau2_,      0, dim->v2tau2     *np*sizeof(double), stream));
      }
    }
  
    if(v3rho3 != NULL){
      assert(v3rho2sigma!=NULL && v3rhosigma2!=NULL && v3sigma3!=NULL);
  
      checkCuda(__FILE__,__LINE__,hipMemsetAsync(v3rho3_,      0, dim->v3rho3     *np*sizeof(double), stream));
      if(is_gga(func->info->family)){
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v3rho2sigma_, 0, dim->v3rho2sigma*np*sizeof(double), stream));
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v3rhosigma2_, 0, dim->v3rhosigma2*np*sizeof(double), stream));
        checkCuda(__FILE__,__LINE__,hipMemsetAsync(v3sigma3_,    0, dim->v3sigma3   *np*sizeof(double), stream));
      }
    }
    /* hipMemset is supposed to be blocking
    stat = hipDeviceSynchronize();
    if (stat != hipSuccess) {
        fprintf(stderr,"Error: mix_init_offload pre-launch: %s\n",hipGetErrorString( stat ));
    }
    else {
        fprintf(stderr,"Success: pre-launch sync\n");
    }
    */
    aux = func->func_aux[ii];
    if (aux->n_func_aux > 0) {
      fprintf(stderr,"Nested multi-term functionals in: %s\n",
              xc_functional_get_name(func->info->number));
      exit(1);
    }
    switch(aux->info->family){
    case XC_FAMILY_LDA:
      xc_lda_offload(aux, np, rho, zk_, vrho_, v2rho2_, NULL, stream);
      break;
    case XC_FAMILY_GGA:
      xc_gga_offload(aux, np, rho, sigma, zk_, vrho_, vsigma_,
             v2rho2_, v2rhosigma_, v2sigma2_,
             v3rho3_, v3rho2sigma_, v3rhosigma2_, v3sigma3_, stream);
      break;
    case XC_FAMILY_MGGA:
      xc_mgga_offload(aux, np, rho, sigma, lapl, tau,
              zk_,
              vrho_, vsigma_, vlapl_, vtau_,
              v2rho2_, v2rhosigma_, v2rholapl_, v2rhotau_,
              v2sigma2_, v2sigmalapl_, v2sigmatau_,
              v2lapl2_, v2lapltau_,
              v2tau2_,
              v3rho3_, v3rho2sigma_, v3rho2lapl_, v3rho2tau_,
              v3rhosigma2_, v3rhosigmalapl_, v3rhosigmatau_,
              v3rholapl2_, v3rholapltau_,
              v3rhotau2_,
              v3sigma3_, v3sigma2lapl_, v3sigma2tau_,
              v3sigmalapl2_, v3sigmalapltau_,
              v3sigmatau2_,
              v3lapl3_, v3lapl2tau_,
              v3lapltau2_,
              v3tau3_,
              stream);
      break;
    }
    //stat = hipDeviceSynchronize();
    //if (stat != hipSuccess) {
    //    fprintf(stderr,"Error: mix_init_offload post-launch: %s\n",hipGetErrorString( stat ));
    //}
    //else {
    //    fprintf(stderr,"Success: post-launch sync\n");
    //}

    /* Sanity checks */
    if(is_gga(aux->info->family))
      assert(is_gga(func->info->family));
    if(is_mgga(aux->info->family) && !is_mgga(func->info->family))
      assert(is_mgga(func->info->family));
    if(aux->info->flags & XC_FLAGS_NEEDS_LAPLACIAN)
      assert(func->info->flags & XC_FLAGS_NEEDS_LAPLACIAN);

    if(zk != NULL) {
      checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->zk,device_funcs_coef+ii,
                                                zk_,1,zk,1));
    }

    if(vrho != NULL) {
      checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->vrho,device_funcs_coef+ii,vrho_,1,vrho,1));

      if(is_gga(aux->info->family)) {
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->vsigma,device_funcs_coef+ii,vsigma_,1,vsigma,1));
      }

      if(is_mgga(aux->info->family)) {
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->vtau,device_funcs_coef+ii,vtau_,1,vtau,1));
        if(aux->info->flags & XC_FLAGS_NEEDS_LAPLACIAN) {
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->lapl,device_funcs_coef+ii,vlapl_,1,vlapl,1));
        }
      }
    }

    if(v2rho2 != NULL){
      checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v2rho2,device_funcs_coef+ii,v2rho2_,1,v2rho2,1));

      if(is_gga(aux->info->family)) {
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v2rhosigma,device_funcs_coef+ii,v2rhosigma_,1,v2rhosigma,1));
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v2sigma2,device_funcs_coef+ii,v2sigma2_,1,v2sigma2,1));
      }

      if(is_mgga(aux->info->family)) {
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v2rhotau,device_funcs_coef+ii,v2rhotau_,1,v2rhotau,1));
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v2sigmatau,device_funcs_coef+ii,v2sigmatau_,1,v2sigmatau,1));
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v2tau2,device_funcs_coef+ii,v2tau2_,1,v2tau2,1));
        if(aux->info->flags & XC_FLAGS_NEEDS_LAPLACIAN) {
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v2rholapl,device_funcs_coef+ii,v2rholapl_,1,v2rholapl,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v2lapl2,device_funcs_coef+ii,v2lapl2_,1,v2lapl2,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v2sigmalapl,device_funcs_coef+ii,v2sigmalapl_,1,v2sigmalapl,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v2lapltau,device_funcs_coef+ii,v2lapltau_,1,v2lapltau,1));
        }
      }
    }

    if(v3rho3 != NULL){
      checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3rho3,device_funcs_coef+ii,v3rho3_,1,v3rho3,1));
      
      if(is_gga(aux->info->family)) {
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3rho2sigma,device_funcs_coef+ii,v3rho2sigma_,1,v3rho2sigma,1));
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3rhosigma2,device_funcs_coef+ii,v3rhosigma2_,1,v3rhosigma2,1));
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3sigma3,device_funcs_coef+ii,v3sigma3_,1,v3sigma3,1));
      }
      if(is_mgga(aux->info->family)) {
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3rho2tau,device_funcs_coef+ii,v3rho2tau_,1,v3rho2tau,1));
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3rhotau2,device_funcs_coef+ii,v3rhotau2_,1,v3rhotau2,1));
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3sigma2tau,device_funcs_coef+ii,v3sigma2tau_,1,v3sigma2tau,1));
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3sigmatau2,device_funcs_coef+ii,v3sigmatau2_,1,v3sigmatau2,1));
        checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3tau3,device_funcs_coef+ii,v3tau3_,1,v3tau3,1));

        if(aux->info->flags & XC_FLAGS_NEEDS_LAPLACIAN) {
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3rho2lapl,device_funcs_coef+ii,v3rho2lapl_,1,v3rho2lapl,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3rholapl2,device_funcs_coef+ii,v3rholapl2_,1,v3rholapl2,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3rholapltau,device_funcs_coef+ii,v3rholapltau_,1,v3rholapltau,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3sigma2lapl,device_funcs_coef+ii,v3sigma2lapl_,1,v3sigma2lapl,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3sigmalapl2,device_funcs_coef+ii,v3sigmalapl2_,1,v3sigmalapl2,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3sigmalapltau,device_funcs_coef+ii,v3sigmalapltau_,1,v3sigmalapltau,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3lapl3,device_funcs_coef+ii,v3lapl3_,1,v3lapl3,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3lapl2tau,device_funcs_coef+ii,v3lapl2tau_,1,v3lapl2tau,1));
          checkCublas(__FILE__,__LINE__,hipblasDaxpy(cublas_handle,np*dim->v3lapltau2,device_funcs_coef+ii,v3lapltau2_,1,v3lapltau2,1));
        }
      }  
    }
    //stat = hipDeviceSynchronize();
    //if (stat != hipSuccess) {
    //    fprintf(stderr,"Error: mix_init_offload post-daxpy: %s\n",hipGetErrorString( stat ));
    //}
    //else {
    //    fprintf(stderr,"Success: post-daxpy sync\n");
    //}
  }
  checkCublas(__FILE__,__LINE__,hipblasSetStream(cublas_handle,NULL));

}
