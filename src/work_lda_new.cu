#include "hip/hip_runtime.h"
/*
 Copyright (C) 2006-2018 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

/**
 * @file work_lda.cu
 * @brief This file is to be included in LDA functionals. As often these
 *        functionals are written as a function of rs and zeta, this
 *        routine performs the necessary conversions between this and a functional
 *        of rho.
 */
#ifdef __HIPCC__

#include "xc_func_data.cuh"

#ifdef XC_NO_EXC
#define OUT_PARAMS vrho, v2rho2, v3rho3
#else
#define OUT_PARAMS zk, vrho, v2rho2, v3rho3
#endif

/**
 * @param[in,out] func_type: pointer to functional structure
 */
__global__ static void 
work_lda_offload(const XC(func_type) *p, int np, const double *rho, 
                 double *zk, double *vrho, double *v2rho2, double *v3rho3)
{
    const xc_dimensions *dim = &(func->dim);
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    double *rho_    = NULL;
    double *zk_     = NULL;
    double *vrho_   = NULL;
    double *v2rho2_ = NULL;
    double *v3rho3_ = NULL;
    rho_ = rho+tid*dim->rho;
    if (zk     != NULL) zk_     = zk+tid*dim->zk;
    if (vrho   != NULL) vrho_   = vrho+tid*dim->vrho;
    if (v2rho2 != NULL) v2rho2_ = v2rho2+tid*dim->v2rho2;
    if (v3rho3 != NULL) v3rho3_ = v3rho3+tid*dim->v3rho3;
    work_lda(xc_func_data[p->func_rank],1,rho_,zk_,vrho_,v2rho2_,v3rho3_);
}

#endif
