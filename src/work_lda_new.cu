#include "hip/hip_runtime.h"
/*
 Copyright (C) 2006-2018 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

/**
 * @file work_lda.cu
 * @brief This file is to be included in LDA functionals. As often these
 *        functionals are written as a function of rs and zeta, this
 *        routine performs the necessary conversions between this and a functional
 *        of rho.
 */
#ifdef __HIPCC__

#include <stdio.h>
extern xc_func_type *xc_func_data_device;

#ifdef XC_NO_EXC
#define OUT_PARAMS vrho, v2rho2, v3rho3
#else
#define OUT_PARAMS zk, vrho, v2rho2, v3rho3
#endif

/**
 * @param[in,out] func_type: pointer to functional structure
 */
__global__ static void 
work_lda_device(const XC(func_type) *p,
                int dim_rho, int dim_zk, int dim_vrho, int dim_v2rho2, int dim_v3rho3,
                int np, const double *rho, 
                double *zk, double *vrho, double *v2rho2, double *v3rho3)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if( tid < np ) {
        const double *rho_    = NULL;
        double       *zk_     = NULL;
        double       *vrho_   = NULL;
        double       *v2rho2_ = NULL;
        double       *v3rho3_ = NULL;
        rho_ = rho+tid*dim_rho;
        if (zk     != NULL) zk_     = zk+tid*dim_zk;
        if (vrho   != NULL) vrho_   = vrho+tid*dim_vrho;
        if (v2rho2 != NULL) v2rho2_ = v2rho2+tid*dim_v2rho2;
        if (v3rho3 != NULL) v3rho3_ = v3rho3+tid*dim_v3rho3;
        work_lda(p,1,rho_,zk_,vrho_,v2rho2_,v3rho3_);
    }
}

static void 
work_lda_offload(const XC(func_type) *p, int np, const double *rho, 
                 double *zk, double *vrho, double *v2rho2, double *v3rho3)
{
    const xc_dimensions *dim = &(p->dim);
    work_lda_device<<<std::ceil(np/1024.),1024>>>
                   (xc_func_data_device+p->func_rank,
                    dim->rho,dim->zk,dim->vrho,dim->v2rho2,dim->v3rho3,
                    np,rho,zk,vrho,v2rho2,v3rho3);
    hipError_t stat = hipGetLastError();
    if (stat != hipSuccess) {
        printf("Launch work_lda_device: %s\n",hipGetErrorString( stat ));
    }
}

#endif
