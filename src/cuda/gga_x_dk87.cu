/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include "util.h"
#include "dvc_util.h"

#define XC_GGA_X_DK87_R1      111 /* dePristo & Kress 87 (version R1)               */
#define XC_GGA_X_DK87_R2      112 /* dePristo & Kress 87 (version R2)               */

#pragma omp declare target

typedef struct {
  double a1, b1, alpha;
} gga_x_dk87_params;

DEVICE
static const gga_x_dk87_params dvc_par_dk87_r1 = {
  0.861504, 0.044286, 1.0
};

DEVICE
static const gga_x_dk87_params dvc_par_dk87_r2 = {
  0.861213, 0.042076, 0.98
};

DEVICE
static void 
dvc_gga_x_dk87_init(xc_func_type *p)
{
  gga_x_dk87_params *params;

  assert(p!=NULL && p->params == NULL);
  p->params = malloc(sizeof(gga_x_dk87_params));
  params = (gga_x_dk87_params *) (p->params);

  switch(p->info->number){
  case XC_GGA_X_DK87_R1: 
    memcpy(params, &dvc_par_dk87_r1, sizeof(gga_x_dk87_params));
    break;
  case XC_GGA_X_DK87_R2:
    memcpy(params, &dvc_par_dk87_r2, sizeof(gga_x_dk87_params));
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in gga_x_dk87\n");
    exit(1);
    #endif
    break;
  }
}

#include "maple2c/gga_exc/gga_x_dk87.c"
#include "work_gga_new.cu"

DEVICE
const xc_func_info_type dvc_xc_func_info_gga_x_dk87_r1 = {
  XC_GGA_X_DK87_R1,
  XC_EXCHANGE,
  "dePristo & Kress 87 version R1",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_DePristo1987_1425, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-24,
  0, NULL, NULL,
  dvc_gga_x_dk87_init, NULL, 
  NULL, dvc_work_gga, NULL
};

DEVICE
const xc_func_info_type dvc_xc_func_info_gga_x_dk87_r2 = {
  XC_GGA_X_DK87_R2,
  XC_EXCHANGE,
  "dePristo & Kress 87 version R2",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_DePristo1987_1425, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-24,
  0, NULL, NULL,
  dvc_gga_x_dk87_init, NULL, 
  NULL, dvc_work_gga, NULL
};

#pragma omp end declare target
