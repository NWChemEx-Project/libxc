/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"
#include "dvc_util.h"

/************************************************************************
Correlation functional by Pittalis, Rasanen & Marques for the 2D electron gas
************************************************************************/

/* TODO: convert this to an (rs, zeta) expression */

#define XC_LDA_C_2D_PRM  16   /* Pittalis, Rasanen & Marques correlation in 2D */

#pragma omp declare target

typedef struct{
  double N;
  double c;
} lda_c_2d_prm_params;

/* Initialization */
DEVICE static void
dvc_lda_c_2d_prm_init(xc_func_type *p)
{
  assert(p != NULL && p->params == NULL);

  p->params = malloc(sizeof(lda_c_2d_prm_params));
}

#include "maple2c/lda_exc/lda_c_2d_prm.c"
#include "work_lda_new.cu"

DEVICE static const func_params_type dvc_ext_params[] = {
  {"N", 2.0, "Number of electrons"},
};

DEVICE static void 
dvc_set_ext_params(xc_func_type *p, const double *ext_params)
{
  static double prm_q = 3.9274; /* 2.258 */
  lda_c_2d_prm_params *params;
  double ff;

  assert(p != NULL && p->params != NULL);
  params = (lda_c_2d_prm_params *) (p->params);

  params->N = dvc_get_ext_param(p->info->ext_params, ext_params, 0);

  if(params->N <= 1.0){
    #ifndef __HIPCC__
    fprintf(stderr, "PRM functional cannot be used for N_electrons <= 1\n");
    exit(1);
    #endif
  }

  params->c = M_PI/(2.0*(params->N - 1.0)*prm_q*prm_q); /* Eq. (13) */
}

extern DEVICE const xc_func_info_type dvc_xc_func_info_lda_c_2d_prm = {
  XC_LDA_C_2D_PRM,
  XC_CORRELATION,
  "PRM (for 2D systems)",
  XC_FAMILY_LDA,
  {&dvc_xc_ref_Pittalis2008_195322, NULL, NULL, NULL, NULL},
  XC_FLAGS_2D | XC_FLAGS_I_HAVE_ALL,
  1e-32,
  1, dvc_ext_params, dvc_set_ext_params,
  dvc_lda_c_2d_prm_init, NULL,
  dvc_work_lda, NULL, NULL
};
#pragma omp end declare target
