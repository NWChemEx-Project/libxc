/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"
#include "dvc_util.h"

/************************************************************************
 Implements Tao, Perdew, Staroverov & Scuseria 
   meta-Generalized Gradient Approximation.

  Exchange part
************************************************************************/

#define XC_MGGA_X_TPSS          202 /* Tao, Perdew, Staroverov & Scuseria exchange */
#define XC_MGGA_X_MODTPSS       245 /* Modified Tao, Perdew, Staroverov & Scuseria exchange */
#define XC_MGGA_X_REVTPSS       212 /* revised Tao, Perdew, Staroverov & Scuseria exchange */
#define XC_MGGA_X_BLOC          244 /* functional with balanced localization */

#pragma omp declare target

typedef struct{
  double b, c, e, kappa, mu;
  double BLOC_a, BLOC_b;
} mgga_x_tpss_params;

DEVICE static const mgga_x_tpss_params dvc_par_modtpss = {0.40, 1.38496, 1.37, 0.804, 0.252, 2.0, 0.0};
DEVICE static const mgga_x_tpss_params dvc_par_revtpss = {0.40, 2.35203946, 2.16769874, 0.804, 0.14, 3.0, 0.0};
DEVICE static const mgga_x_tpss_params dvc_par_bloc    = {0.40, 1.59096, 1.537, 0.804, 0.21951, 4.0, -3.3};

DEVICE static void 
dvc_mgga_x_tpss_init(xc_func_type *p)
{
  mgga_x_tpss_params *params;

  assert(p!=NULL && p->params == NULL);
  p->params = malloc(sizeof(mgga_x_tpss_params));
  params = (mgga_x_tpss_params *)p->params;

  switch(p->info->number){
  case XC_MGGA_X_TPSS:
    /* default set by set_ext_params */
    break;
  case XC_MGGA_X_MODTPSS:
    memcpy(params, &dvc_par_modtpss, sizeof(mgga_x_tpss_params));
    break;
  case XC_MGGA_X_REVTPSS:
    memcpy(params, &dvc_par_revtpss, sizeof(mgga_x_tpss_params));
    break;
  case XC_MGGA_X_BLOC:
    memcpy(params, &dvc_par_bloc, sizeof(mgga_x_tpss_params));
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in mgga_x_tpss\n");
    exit(1);
    #endif
    break;
  }
}

DEVICE static const func_params_type dvc_ext_params[] = {
  {"_b",      0.40,    "b"},
  {"_c",      1.59096, "c"},
  {"_e",      1.537,   "e"},
  {"_kappa",  0.8040,  "Asymptotic value of the enhancement function"},
  {"_mu",     0.21951, "Coefficient of the 2nd order expansion"},
  {"_BLOC_a", 2.0,     "BLOC_a"},
  {"_BLOC_b", 0.0,     "BLOC_b"},
};

DEVICE static void 
dvc_set_ext_params(xc_func_type *p, const double *ext_params)
{
  mgga_x_tpss_params *params;

  assert(p != NULL && p->params != NULL);
  params = (mgga_x_tpss_params *) (p->params);

  params->b      = dvc_get_ext_param(p->info->ext_params, ext_params, 0);
  params->c      = dvc_get_ext_param(p->info->ext_params, ext_params, 1);
  params->e      = dvc_get_ext_param(p->info->ext_params, ext_params, 2);
  params->kappa  = dvc_get_ext_param(p->info->ext_params, ext_params, 3);
  params->mu     = dvc_get_ext_param(p->info->ext_params, ext_params, 4);
  params->BLOC_a = dvc_get_ext_param(p->info->ext_params, ext_params, 5);
  params->BLOC_b = dvc_get_ext_param(p->info->ext_params, ext_params, 6);
}

#include "maple2c/mgga_exc/mgga_x_tpss.c"
#include "work_mgga_new.cu"

DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_tpss = {
  XC_MGGA_X_TPSS,
  XC_EXCHANGE,
  "Tao, Perdew, Staroverov & Scuseria",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Tao2003_146401, &dvc_xc_ref_Perdew2004_6898, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-23,
  7, dvc_ext_params, dvc_set_ext_params,
  dvc_mgga_x_tpss_init, NULL, 
  NULL, NULL, dvc_work_mgga,
};

DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_modtpss = {
  XC_MGGA_X_MODTPSS,
  XC_EXCHANGE,
  "Modified Tao, Perdew, Staroverov & Scuseria",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Perdew2007_042506, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-23,
  0, NULL, NULL,
  dvc_mgga_x_tpss_init, NULL, 
  NULL, NULL, dvc_work_mgga,
};

DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_revtpss = {
  XC_MGGA_X_REVTPSS,
  XC_EXCHANGE,
  "revised Tao, Perdew, Staroverov & Scuseria",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Perdew2009_026403, &dvc_xc_ref_Perdew2009_026403_err, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-23,
  0, NULL, NULL,
  dvc_mgga_x_tpss_init, NULL, 
  NULL, NULL, dvc_work_mgga,
};

DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_bloc = {
  XC_MGGA_X_BLOC,
  XC_EXCHANGE,
  "functional with balanced localization",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Constantin2013_2256, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-23,
  0, NULL, NULL,
  dvc_mgga_x_tpss_init, NULL, 
  NULL, NULL, dvc_work_mgga,
};

#pragma omp end declare target
