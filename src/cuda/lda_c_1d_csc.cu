#include "hip/hip_runtime.h"
/*
 Copyright (C) 2006-2009 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include "util.h"
#include "dvc_util.h"

#define XC_LDA_C_1D_CSC          18 /* Casula, Sorella, and Senatore 1D correlation     */

#pragma omp declare target
typedef struct{
  int interaction;  /* 0: exponentially screened; 1: soft-Coulomb */
  double bb;         /* screening parameter */

  const double *para, *ferro;
} lda_c_1d_csc_params;

DEVICE static const double dvc_par_para[][10] = { /* paramagnetic */
  /* 0:A    1:B   2:C    3:D    4:E  5:n1   6:n2  7:alpha  8:beta  9:m */
  {  4.66,  0.0,  2.092, 3.735, 0.0, 1.379, 2.0, 23.63,  109.9,    1.837}, /* exponentially screened interaction */
  {  9.5,   0.0,  1.85,  5.64,  0.0, 0.882, 2.0,  5.346,   6.69,   3.110},
  { 16.40,  0.0,  2.90,  6.235, 0.0, 0.908, 2.0,  3.323,   2.23,   3.368},
  { 22.53,  0.0,  2.09,  7.363, 0.0, 0.906, 2.0,  2.029,   0.394,  4.070},
  { 32.1,   0.0,  3.77,  7.576, 0.0, 0.941, 2.0,  1.63,    0.198,  4.086},
  {110.5,   0.0,  7.90,  8.37,  0.0, 1.287, 2.0,  1.399,   0.0481, 4.260},
  {413.0,   0.0, 10.8,   7.99,  0.0, 1.549, 2.0,  1.308,   0.0120, 4.165},
  { 7.40, 1.120, 1.890, 0.0964,  0.0250,   2.0, 3.0, 2.431, 0.0142, 2.922}, /* soft-Coulomb interaction */
  {18.40, 0.0,   7.501, 0.10185, 0.012827, 2.0, 3.0, 1.511, 0.258,  4.424}
};

DEVICE static const double dvc_par_ferro[][10] = { /* ferromagnetic */
  { 5.24, 0.0,   1.568, 0.12856, 0.003201, 2.0, 3.0, 0.0538, 1.56e-5, 2.958}
};

DEVICE static void 
dvc_lda_c_1d_csc_init(xc_func_type *p)
{
  assert(p != NULL && p->params == NULL);
  p->params = malloc(sizeof(lda_c_1d_csc_params));
}

#include "maple2c/lda_exc/lda_c_1d_csc.c"
#include "work_lda_new.cu"

DEVICE static const func_params_type dvc_ext_params[] = {
  {"interaction",  1, "0 (exponentially screened) | 1 (soft-Coulomb)"},
  {"beta", 1.0, "Screening parameter"}
};

DEVICE static void 
dvc_set_ext_params(xc_func_type *p, const double *ext_params)
{
  lda_c_1d_csc_params *params;

  assert(p != NULL && p->params != NULL);
  params = (lda_c_1d_csc_params *)(p->params);

  params->interaction = (int)round(dvc_get_ext_param(p->info->ext_params, ext_params, 0));
  params->bb = dvc_get_ext_param(p->info->ext_params, ext_params, 1);

  params->para = params->ferro = NULL;

  if(params->interaction == 0){
    if      (params->bb == 0.1){
      params->para  = dvc_par_para[0];
      params->ferro = dvc_par_para[0];
    }else if(params->bb == 0.3){
      params->para  = dvc_par_para[1];
      params->ferro = dvc_par_para[1];
    }else if(params->bb == 0.5){
      params->para  = dvc_par_para[2];
      params->ferro = dvc_par_para[2];
    }else if(params->bb == 0.75){
      params->para  = dvc_par_para[3];
      params->ferro = dvc_par_para[3];
    }else if(params->bb == 1.0){
      params->para  = dvc_par_para[4];
      params->ferro = dvc_par_para[4];
    }else if(params->bb == 2.0){
      params->para  = dvc_par_para[5];
      params->ferro = dvc_par_para[5];
    }else if(params->bb == 4.0){
      params->para  = dvc_par_para[6];
      params->ferro = dvc_par_para[6];
    }
  }else if(params->interaction == 1){
    if     (params->bb == 0.5){
      params->para  = dvc_par_para[7];
      params->ferro = dvc_par_para[7];
    }else if(params->bb == 1.0){
      params->para  = dvc_par_para[8];
      params->ferro = dvc_par_ferro[0];
    }
  }

  if(params->para == NULL){
    #ifndef __HIPCC__
    fprintf(stderr, "Invalid value of parameters (inter,b) = (%d,%f) in lda_c_1d_csc_set_params", 
	    params->interaction, params->bb);
    exit(1);
    #endif
  }
}


extern DEVICE const xc_func_info_type dvc_xc_func_info_lda_c_1d_csc = {
  XC_LDA_C_1D_CSC,
  XC_CORRELATION,
  "Casula, Sorella & Senatore",
  XC_FAMILY_LDA,
  {&dvc_xc_ref_Casula2006_245427, NULL, NULL, NULL, NULL},
  XC_FLAGS_1D |  XC_FLAGS_I_HAVE_ALL,
  5e-26,
  2, dvc_ext_params, dvc_set_ext_params,
  dvc_lda_c_1d_csc_init, NULL,
  dvc_work_lda, NULL, NULL
};
#pragma omp end declare target
