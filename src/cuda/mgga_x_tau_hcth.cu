/*
 Copyright (C) 2008 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"
#include "dvc_util.h"

#define XC_MGGA_X_TAU_HCTH        205 /* tau-HCTH from Boese and Handy */
#define XC_HYB_MGGA_X_BMK         279 /* Boese-Martin for kinetics     */
#define XC_HYB_MGGA_X_TAU_HCTH    282 /* Hybrid version of tau-HCTH    */

#pragma omp declare target

DEVICE const double dvc_tHCTH_cx_local [4] = {1.10734, -1.0534, 6.3491, -2.5531};
DEVICE const double dvc_tHCTH_cx_nlocal[4] = {0.00110, -0.3041, 6.9543, -0.7235};

DEVICE const double dvc_BMK_cx_local [4] = { 0.474302, 2.77701, -11.4230, 11.7167};
DEVICE const double dvc_BMK_cx_nlocal[4] = {-0.192212, 4.73936, -26.6188, 22.4891};

DEVICE const double dvc_hyb_tHCTH_cx_local [4] = { 0.86735,  0.3008, 1.2208,   0.1574};
DEVICE const double dvc_hyb_tHCTH_cx_nlocal[4] = {-0.00230, -0.2849, 5.4146, -10.909};

typedef struct{
  const double *cx_local;
  const double *cx_nlocal;
} mgga_x_tau_hcth_params;


DEVICE static void 
dvc_mgga_x_tau_hcth_init(xc_func_type *p)
{
  mgga_x_tau_hcth_params *params;

  assert(p != NULL);
  assert(p->params == NULL);

  p->params = malloc(sizeof(mgga_x_tau_hcth_params));
  params = (mgga_x_tau_hcth_params *)(p->params);

  switch(p->info->number){
  case XC_MGGA_X_TAU_HCTH:
    params->cx_local  = dvc_tHCTH_cx_local;
    params->cx_nlocal = dvc_tHCTH_cx_nlocal;
    break;
  case XC_HYB_MGGA_X_BMK:
    p->cam_alpha = 0.42;
    params->cx_local  = dvc_BMK_cx_local;
    params->cx_nlocal = dvc_BMK_cx_nlocal;
    break;
  case XC_HYB_MGGA_X_TAU_HCTH:
    p->cam_alpha = 0.15;
    params->cx_local  = dvc_hyb_tHCTH_cx_local;
    params->cx_nlocal = dvc_hyb_tHCTH_cx_nlocal;
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in mgga_tau_hcth\n");
    exit(1);
    #endif
    break;
  }
}

#include "maple2c/mgga_exc/mgga_x_tau_hcth.c"
#include "work_mgga_new.cu"

DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_tau_hcth = {
  XC_MGGA_X_TAU_HCTH,
  XC_EXCHANGE,
  "tau-HCTH from Boese and Handy",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Boese2002_9559, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1.0e-23,
  0, NULL, NULL,
  dvc_mgga_x_tau_hcth_init, NULL, 
  NULL, NULL, dvc_work_mgga,
};

DEVICE const xc_func_info_type dvc_xc_func_info_hyb_mgga_x_bmk = {
  XC_HYB_MGGA_X_BMK,
  XC_EXCHANGE,
  "Boese-Martin for kinetics",
  XC_FAMILY_HYB_MGGA,
  {&dvc_xc_ref_Boese2004_3405, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1.0e-32,
  0, NULL, NULL,
  dvc_mgga_x_tau_hcth_init, NULL, 
  NULL, NULL, dvc_work_mgga,
};

DEVICE const xc_func_info_type dvc_xc_func_info_hyb_mgga_x_tau_hcth = {
  XC_HYB_MGGA_X_TAU_HCTH,
  XC_EXCHANGE,
  "Hybrid version of tau-HCTH",
  XC_FAMILY_HYB_MGGA,
  {&dvc_xc_ref_Boese2002_9559, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1.0e-32,
  0, NULL, NULL,
  dvc_mgga_x_tau_hcth_init,  NULL, 
  NULL, NULL, dvc_work_mgga,
};

#pragma omp end declare target
