#include "hip/hip_runtime.h"
/*
 Copyright (C) 2017 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include "util.h"
#include "dvc_util.h"

#define XC_LDA_X_ERF   546   /* Attenuated exchange LDA (erf) */

#pragma omp declare target

DEVICE static const func_params_type dvc_ext_params[] = {
  {"omega",  0.3, "screening parameter"},
};

DEVICE static void 
dvc_set_ext_params(xc_func_type *p, const double *ext_params)
{
  assert(p != NULL);

  p->cam_omega = dvc_get_ext_param(p->info->ext_params, ext_params, 0);
}


#include "maple2c/lda_exc/lda_x_erf.c"
#include "work_lda_new.cu"

extern DEVICE const xc_func_info_type dvc_xc_func_info_lda_x_erf = {
  XC_LDA_X_ERF,
  XC_EXCHANGE,
  "Attenuated exchange LDA (erf)",
  XC_FAMILY_LDA,
  {&dvc_xc_ref_Gill1996_1005, &dvc_xc_ref_Toulouse2004_1047, &dvc_xc_ref_Tawada2004_8425, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-13,
  1, dvc_ext_params, dvc_set_ext_params,
  NULL, NULL, 
  dvc_work_lda, NULL, NULL
};

#pragma omp end declare target
