#include "hip/hip_runtime.h"
/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include "xc.h"
#include "util.h"
#include "dvc_util.h"
#include "funcs_key.h"
//ifdef __HIPCC__
//undef max
//undef min
//include <string>
//else
#ifndef __HIPCC__
#include <string.h>
#endif
#ifdef _MSC_VER
#define strcasecmp _stricmp
#define strncasecmp _strnicmp
#else
#include <strings.h>
#endif

#pragma omp declare target
/* This does not work in C++ as the language insists that the
 * size of the arrays be known. It does not let you to just
 * get a pointer to an array. Hence we replace this code 
 * with the code that actually defines the arrays.

extern DEVICE xc_func_info_type *xc_lda_known_funct[]; 
extern DEVICE xc_func_info_type *xc_gga_known_funct[];
extern DEVICE xc_func_info_type *xc_hyb_gga_known_funct[];
extern DEVICE xc_func_info_type *xc_mgga_known_funct[];
extern DEVICE xc_func_info_type *xc_hyb_mgga_known_funct[];
*/
#include "funcs_lda.h"
#include "funcs_gga.h"
#include "funcs_hyb_gga.h"
#include "funcs_mgga.h"
#include "funcs_hyb_mgga.h"


/*------------------------------------------------------*/
DEVICE int dvc_xc_functional_get_number(const char *name)
{
  int ii;
  int key=-1;
  const char *p;

  /* Does name begin with xc_? */
  if(dvc_strncasecmp(name,"XC_",3) == 0) {
    p=name+3;
  } else {
    p=name;
  }

  for(ii=0;;ii++){
    if(dvc_xc_functional_keys[ii].number == -1)
      break;
    if(dvc_strcasecmp(dvc_xc_functional_keys[ii].name, p) == 0){
      key = dvc_xc_functional_keys[ii].number;
      break;
    }
  }
  
  return key;
}


/*------------------------------------------------------*/
DEVICE char *dvc_xc_functional_get_name(int number)
{
  int ii;
  char *p;

  for(ii=0;;ii++){
    if(dvc_xc_functional_keys[ii].number == -1)
      return NULL;
    if(dvc_xc_functional_keys[ii].number == number) {
      /* return duplicated: caller has the responsibility to dealloc string.
         Do this the old way since strdup and strndup aren't C standard. */
      p=(char*)malloc(dvc_strlen(dvc_xc_functional_keys[ii].name)+1);
      dvc_strcpy(p,dvc_xc_functional_keys[ii].name);
      return p;
    }
  }
}


/*------------------------------------------------------*/
DEVICE int dvc_xc_family_from_id(int id, int *family, int *number)
{
  int ii;

  /* first let us check if it is an LDA */
  for(ii=0; dvc_xc_lda_known_funct[ii]!=NULL; ii++){
    if(dvc_xc_lda_known_funct[ii]->number == id){
      if(family != NULL) *family = XC_FAMILY_LDA;
      if(number != NULL) *number = ii;
      return XC_FAMILY_LDA;
    }
  }

  /* or is it a GGA? */
  for(ii=0; dvc_xc_gga_known_funct[ii]!=NULL; ii++){
    if(dvc_xc_gga_known_funct[ii]->number == id){
      if(family != NULL) *family = XC_FAMILY_GGA;
      if(number != NULL) *number = ii;
      return XC_FAMILY_GGA;
    }
  }

  /* or is it a hybrid GGA? */
  for(ii=0; dvc_xc_hyb_gga_known_funct[ii]!=NULL; ii++){
    if(dvc_xc_hyb_gga_known_funct[ii]->number == id){
      if(family != NULL) *family = XC_FAMILY_HYB_GGA;
      if(number != NULL) *number = ii;
      return XC_FAMILY_HYB_GGA;
    }
  }

  /* or is it a meta GGA? */
  for(ii=0; dvc_xc_mgga_known_funct[ii]!=NULL; ii++){
    if(dvc_xc_mgga_known_funct[ii]->number == id){
      if(family != NULL) *family = XC_FAMILY_MGGA;
      if(number != NULL) *number = ii;
      return XC_FAMILY_MGGA;
    }
  }

  /* or is it a hybrid meta GGA? */
  for(ii=0; dvc_xc_hyb_mgga_known_funct[ii]!=NULL; ii++){
    if(dvc_xc_hyb_mgga_known_funct[ii]->number == id){
      if(family != NULL) *family = XC_FAMILY_HYB_MGGA;
      if(number != NULL) *number = ii;
      return XC_FAMILY_HYB_MGGA;
    }
  }

  return XC_FAMILY_UNKNOWN;
}

/*------------------------------------------------------*/
DEVICE int dvc_xc_number_of_functionals()
{
  int num;

  for(num=0;;num++){
    if(dvc_xc_functional_keys[num].number == -1)
      return num;
  }

  #ifndef __HIPCC__
  fprintf(stderr, "Internal error in functionals.c\n");
  exit(1);
  #endif
}

DEVICE int dvc_xc_maximum_name_length()
{
  int i, N, maxlen, tmp;

  N=dvc_xc_number_of_functionals();

  maxlen=0;
  for(i=0;i<N;i++){
    tmp=dvc_strlen(dvc_xc_functional_keys[i].name);
    if(tmp > maxlen) maxlen=tmp;
  }

  return maxlen;
}

/*------------------------------------------------------*/
DEVICE void dvc_xc_available_functional_numbers(int *list)
{
  int ii, N;
  N=dvc_xc_number_of_functionals();
  for(ii=0;ii<N;ii++){
    list[ii]=dvc_xc_functional_keys[ii].number;
  }
}

DEVICE void dvc_xc_available_functional_names(char **list)
{
  int ii, N;

  N=dvc_xc_number_of_functionals();
  for(ii=0;ii<N;ii++) {
    dvc_strcpy(list[ii],dvc_xc_functional_keys[ii].name);
  }
}

/*------------------------------------------------------*/
DEVICE xc_func_type *dvc_xc_func_alloc()
{
  xc_func_type *func;

  func = (xc_func_type *) malloc (sizeof (xc_func_type));
  return func;
}

/*------------------------------------------------------*/
DEVICE int dvc_xc_func_init(xc_func_type *func, int functional, int nspin)
{
  int number;

  assert(func != NULL);
  assert(nspin==XC_UNPOLARIZED || nspin==XC_POLARIZED);

  /* initialize structure */
  func->nspin       = nspin;

  func->params     = NULL;

  func->n_func_aux = 0;
  func->func_aux   = NULL;
  func->mix_coef   = NULL;
  func->cam_omega = func->cam_alpha = func->cam_beta = 0.0;
  func->nlc_b = func->nlc_C = 0.0;

  switch(dvc_xc_family_from_id(functional, NULL, &number)){
  case(XC_FAMILY_LDA):
    func->info = dvc_xc_lda_known_funct[number];
    dvc_internal_counters_set_lda(func->nspin, &(func->dim));
    break;

  case(XC_FAMILY_GGA):
    func->info = dvc_xc_gga_known_funct[number];
    dvc_internal_counters_set_gga(func->nspin, &(func->dim));
    break;

  case(XC_FAMILY_HYB_GGA):
    func->info = dvc_xc_hyb_gga_known_funct[number];
    dvc_internal_counters_set_gga(func->nspin, &(func->dim));
    break;

  case(XC_FAMILY_MGGA):
    func->info = dvc_xc_mgga_known_funct[number];
    dvc_internal_counters_set_mgga(func->nspin, &(func->dim));
    break;

  case(XC_FAMILY_HYB_MGGA):
    func->info = dvc_xc_hyb_mgga_known_funct[number];
    dvc_internal_counters_set_mgga(func->nspin, &(func->dim));
    break;

  default:
    return -2; /* family not found */
  }

  /* see if we need to initialize the functional */
  if(func->info->init != NULL)
    func->info->init(func);

  /* see if we need to initialize the external parameters */
  if(func->info->n_ext_params > 0)
    func->info->set_ext_params(func, NULL);

  func->dens_threshold = func->info->dens_threshold;

  return 0;
}


/*------------------------------------------------------*/
DEVICE void dvc_xc_func_end(xc_func_type *func)
{
  assert(func != NULL && func->info != NULL);

  /* call internal termination routine */
  if(func->info->end != NULL)
    func->info->end(func);

  /* terminate any auxiliary functional */
  if(func->n_func_aux > 0){
    int ii;

    for(ii=0; ii<func->n_func_aux; ii++){
      dvc_xc_func_end(func->func_aux[ii]);
      free(func->func_aux[ii]);
    }
    free(func->func_aux);
    func->n_func_aux = 0;
  }

  if(func->mix_coef != NULL){
    free(func->mix_coef);
    func->mix_coef = NULL;
  }

  /* deallocate any used parameter */
  if(func->params != NULL){
    free(func->params);
    func->params = NULL;
  }

  func->info = NULL;  
}

/*------------------------------------------------------*/
DEVICE void  dvc_xc_func_free(xc_func_type *p)
{
  free(p);
}

/*------------------------------------------------------*/
DEVICE const xc_func_info_type *dvc_xc_func_get_info(const xc_func_type *p)
{
  return p->info;
}

/*------------------------------------------------------*/
DEVICE void dvc_xc_func_set_dens_threshold(xc_func_type *p, double dens_threshold)
{
  int ii;

  p->dens_threshold = dens_threshold;

  for(ii=0; ii<p->n_func_aux; ii++) {
    dvc_xc_func_set_dens_threshold(p->func_aux[ii], dens_threshold);
  }
}

/*------------------------------------------------------*/
DEVICE void dvc_xc_func_set_ext_params(xc_func_type *p, double *ext_params)
{
  assert(p->info->n_ext_params > 0);
  p->info->set_ext_params(p, ext_params);
}

/* returns the mixing coefficient for the hybrid GGAs */
DEVICE double dvc_xc_hyb_exx_coef(const xc_func_type *p)
{
  assert(p!=NULL);
 
  return p->cam_alpha;
}

/* returns the CAM parameters for screened hybrids */
DEVICE void dvc_xc_hyb_cam_coef(const xc_func_type *p, double *omega, double *alpha, double *beta)
{
  assert(p!=NULL);

  *omega = p->cam_omega;
  *alpha = p->cam_alpha;
  *beta  = p->cam_beta;
}

/* returns the NLC parameters */
DEVICE void dvc_xc_nlc_coef(const xc_func_type *p, double *nlc_b, double *nlc_C)
{
  assert(p!=NULL);

  *nlc_b = p->nlc_b;
  *nlc_C = p->nlc_C;
}
#pragma omp end declare target
