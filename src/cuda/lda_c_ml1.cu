/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"
#include "dvc_util.h"

#define XC_LDA_C_ML1    22   /* Modified LSD (version 1) of Proynov and Salahub */
#define XC_LDA_C_ML2    23   /* Modified LSD (version 2) of Proynov and Salahub */

#pragma omp declare target

typedef struct {
  double fc, q;
} lda_c_ml1_params;

DEVICE static void 
dvc_lda_c_ml1_init(xc_func_type *p)
{
  lda_c_ml1_params *params;

  assert(p!=NULL && p->params == NULL);
  p->params = malloc(sizeof(lda_c_ml1_params));
  params = (lda_c_ml1_params *) (p->params);

  switch(p->info->number){
  case XC_LDA_C_ML1:
    params->fc = 0.2026;
    params->q  = 0.084;
    break;
  case XC_LDA_C_ML2:
    params->fc = 0.266;
    params->q  = 0.5;
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in lda_c_ml1\n");
    exit(1);
    #endif
    break;
  }
}

#include "maple2c/lda_exc/lda_c_ml1.c"
#include "work_lda_new.cu"

DEVICE const xc_func_info_type dvc_xc_func_info_lda_c_ml1 = {
  XC_LDA_C_ML1,
  XC_CORRELATION,
  "Modified LSD (version 1) of Proynov and Salahub",
  XC_FAMILY_LDA,
  {&dvc_xc_ref_Proynov1994_7874, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-24,
  0, NULL, NULL,
  dvc_lda_c_ml1_init, NULL,
  dvc_work_lda, NULL, NULL
};

DEVICE const xc_func_info_type dvc_xc_func_info_lda_c_ml2 = {
  XC_LDA_C_ML2,
  XC_CORRELATION,
  "Modified LSD (version 2) of Proynov and Salahub",
  XC_FAMILY_LDA,
  {&dvc_xc_ref_Proynov1994_7874, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-24,
  0, NULL, NULL,
  dvc_lda_c_ml1_init, NULL,
  dvc_work_lda, NULL, NULL
};
#pragma omp end declare target
