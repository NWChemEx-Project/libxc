/*
 Copyright (C) 2016 Susi Lehtola

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"
#include "dvc_util.h"

#define XC_MGGA_X_SCAN          263 /* SCAN exchange of Sun, Ruzsinszky, and Perdew  */
#define XC_HYB_MGGA_X_SCAN0     264 /* SCAN hybrid exchange */
#define XC_MGGA_X_REVSCAN       581 /* revised SCAN */
#define XC_HYB_MGGA_X_REVSCAN0  583 /* revised SCAN hybrid exchange */

#pragma omp declare target

typedef struct{
  double c1, c2, d, k1;
} mgga_x_scan_params;

DEVICE static const mgga_x_scan_params dvc_par_scan = {0.667, 0.8, 1.24, 0.065};
DEVICE static const mgga_x_scan_params dvc_par_revscan = {0.607, 0.7, 1.37, 0.065};

DEVICE static void 
dvc_mgga_x_scan_init(xc_func_type *p)
{
  mgga_x_scan_params *params;

  assert(p!=NULL && p->params == NULL);
  p->params = malloc(sizeof(mgga_x_scan_params));
  params = (mgga_x_scan_params *)p->params;

  switch(p->info->number){
  case XC_MGGA_X_SCAN:
    memcpy(params, &dvc_par_scan, sizeof(mgga_x_scan_params));
    break;
  case XC_MGGA_X_REVSCAN:
    memcpy(params, &dvc_par_revscan, sizeof(mgga_x_scan_params));
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in mgga_x_scan\n");
    exit(1);
    #endif
    break;
  }  
}

#include "maple2c/mgga_exc/mgga_x_scan.c"
#include "work_mgga_new.cu"

DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_scan = {
  XC_MGGA_X_SCAN,
  XC_EXCHANGE,
  "SCAN exchange of Sun, Ruzsinszky, and Perdew",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Sun2015_036402, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-23,
  0, NULL, NULL,
  dvc_mgga_x_scan_init, NULL,
  NULL, NULL, dvc_work_mgga,
};

DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_revscan = {
  XC_MGGA_X_REVSCAN,
  XC_EXCHANGE,
  "revised SCAN",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Mezei2018_2469, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-23,
  0, NULL, NULL,
  dvc_mgga_x_scan_init, NULL,
  NULL, NULL, dvc_work_mgga,
};

DEVICE static void
dvc_hyb_mgga_x_scan0_init(xc_func_type *p)
{
  static int   funcs_id  [1] = {XC_MGGA_X_SCAN};
  static double funcs_coef[1] = {1.0 - 0.25};

  dvc_xc_mix_init(p, 1, funcs_id, funcs_coef);
  p->cam_alpha = 0.25;
}


DEVICE const xc_func_info_type dvc_xc_func_info_hyb_mgga_x_scan0 = {
  XC_HYB_MGGA_X_SCAN0,
  XC_EXCHANGE,
  "SCAN hybrid exchange (SCAN0)",
  XC_FAMILY_HYB_MGGA,
  {&dvc_xc_ref_Hui2016_044114, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-32,
  0, NULL, NULL,
  dvc_hyb_mgga_x_scan0_init, NULL,
  NULL, NULL, NULL /* this is taken care of by the generic routine */
};


DEVICE static void
dvc_hyb_mgga_x_revscan0_init(xc_func_type *p)
{
  static int   funcs_id  [1] = {XC_MGGA_X_REVSCAN};
  static double funcs_coef[1] = {1.0 - 0.25};

  dvc_xc_mix_init(p, 1, funcs_id, funcs_coef);
  p->cam_alpha = 0.25;
}


DEVICE const xc_func_info_type dvc_xc_func_info_hyb_mgga_x_revscan0 = {
  XC_HYB_MGGA_X_REVSCAN0,
  XC_EXCHANGE,
  "revised SCAN hybrid exchange (SCAN0)",
  XC_FAMILY_HYB_MGGA,
  {&dvc_xc_ref_Mezei2018_2469, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-32,
  0, NULL, NULL,
  dvc_hyb_mgga_x_revscan0_init, NULL,
  NULL, NULL, NULL /* this is taken care of by the generic routine */
};

#pragma omp end declare target
