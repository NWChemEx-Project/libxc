/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include "util.h"
#include "dvc_util.h"

#define XC_LDA_C_HL   4   /* Hedin & Lundqvist            */
#define XC_LDA_C_GL   5   /* Gunnarson & Lundqvist        */
#define XC_LDA_C_vBH 17   /* von Barth & Hedin            */

#pragma omp declare target

typedef struct {
  double r[2], c[2];
} lda_c_hl_params;

DEVICE static const lda_c_hl_params dvc_par_hl = { /* HL unpolarized only*/
  {21.0, 21.0}, {0.0225, 0.0225}
};

DEVICE static const lda_c_hl_params dvc_par_gl = {
  {11.4, 15.9}, {0.0333, 0.0203}
};

DEVICE static const lda_c_hl_params dvc_par_vbh = {
  {30.0, 75.0}, {0.0252, 0.0127}
};

DEVICE static void 
dvc_lda_c_hl_init(xc_func_type *p)
{
  lda_c_hl_params *params;

  assert(p!=NULL && p->params == NULL);
  p->params = malloc(sizeof(lda_c_hl_params));
  params = (lda_c_hl_params *) (p->params);

  switch(p->info->number){
  case XC_LDA_C_HL:
    memcpy(params, &dvc_par_hl, sizeof(lda_c_hl_params));
    break;
  case XC_LDA_C_GL:
    memcpy(params, &dvc_par_gl, sizeof(lda_c_hl_params));
    break;
  case XC_LDA_C_vBH:
    memcpy(params, &dvc_par_vbh, sizeof(lda_c_hl_params));
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in lda_c_hl\n");
    exit(1);
    #endif
    break;
  }
}

#include "maple2c/lda_exc/lda_c_hl.c"
#include "work_lda_new.cu"

DEVICE const xc_func_info_type dvc_xc_func_info_lda_c_hl = {
  XC_LDA_C_HL,
  XC_CORRELATION,
  "Hedin & Lundqvist",
  XC_FAMILY_LDA,
  {&dvc_xc_ref_Hedin1971_2064, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-16,
  0, NULL, NULL,
  dvc_lda_c_hl_init, NULL,
  dvc_work_lda, NULL, NULL
};

DEVICE const xc_func_info_type dvc_xc_func_info_lda_c_gl = {
  XC_LDA_C_GL,
  XC_CORRELATION,
  "Gunnarson & Lundqvist",
  XC_FAMILY_LDA,
  {&dvc_xc_ref_Gunnarsson1976_4274, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-17,
  0, NULL, NULL,
  dvc_lda_c_hl_init, NULL,
  dvc_work_lda, NULL, NULL
};

DEVICE const xc_func_info_type dvc_xc_func_info_lda_c_vbh = {
  XC_LDA_C_vBH,
  XC_CORRELATION,
  "von Barth & Hedin",
  XC_FAMILY_LDA,
  {&dvc_xc_ref_vonBarth1972_1629, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-17,
  0, NULL, NULL,
  dvc_lda_c_hl_init, NULL,
  dvc_work_lda, NULL, NULL
};
#pragma omp end declare target
