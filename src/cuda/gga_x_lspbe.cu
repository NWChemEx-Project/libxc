/*
 Copyright (C) 2006-2007 M.A.L. Marques
 Copyright (C) 2018 Susi Lehtola

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include "util.h"
#include "dvc_util.h"

#define XC_GGA_X_LSPBE        168 /* PW91-like exchange with simple analytical form */

#pragma omp declare target

typedef struct{
  double kappa; /* PBE kappa parameter */
  double mu;    /* PBE mu parameter */
  double alpha; /* alpha parameter, solved automatically */
} gga_x_lspbe_params;

DEVICE
static void 
dvc_gga_x_lspbe_init(xc_func_type *p)
{
  gga_x_lspbe_params *params;

  assert(p!=NULL && p->params == NULL);
  p->params = malloc(sizeof(gga_x_lspbe_params));
  params = (gga_x_lspbe_params *) (p->params);
 
  switch(p->info->number){
  case XC_GGA_X_LSPBE:
    /* default set by set_ext_params */
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in gga_x_lspbe\n");
    exit(1);
    #endif
    break;
  }
}

/* PBE: mu = beta*pi^2/3, beta = 0.06672455060314922 */
DEVICE
static const func_params_type dvc_ext_params[] = {
  {"_kappa", 0.8040, "Asymptotic value of the enhancement function"},
  {"_mu",    MU_PBE, "Coefficient of the 2nd order expansion of the full Lspbe functional"},
  {"_alpha", 0.00145165, "Exponent that should satisfy the PW91 criterion"}
};

DEVICE
static void 
dvc_set_ext_params(xc_func_type *p, const double *ext_params)
{
  gga_x_lspbe_params *params;

  assert(p != NULL && p->params != NULL);
  params = (gga_x_lspbe_params *) (p->params);

  params->kappa = dvc_get_ext_param(p->info->ext_params, ext_params, 0);
  params->mu    = dvc_get_ext_param(p->info->ext_params, ext_params, 1);
  params->alpha = dvc_get_ext_param(p->info->ext_params, ext_params, 2);

  /* adapt used mu value to yield wanted mu near origin (eq 9) */
  params-> mu += params->alpha*(1.0 + params->kappa);
}

#include "maple2c/gga_exc/gga_x_lspbe.c"
#include "work_gga_new.cu"

extern DEVICE
const xc_func_info_type dvc_xc_func_info_gga_x_lspbe = {
  XC_GGA_X_LSPBE,
  XC_EXCHANGE,
  "lsPBE, a PW91-like modification of PBE exchange",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_PachecoKato2016_268, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-32,
  3, dvc_ext_params, dvc_set_ext_params,
  dvc_gga_x_lspbe_init, NULL, 
  NULL, dvc_work_gga, NULL
};

#pragma omp end declare target
