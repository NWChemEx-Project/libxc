/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"
#include "dvc_util.h"

#define XC_GGA_C_WI0 153 /* Wilson & Ivanov initial version */
#define XC_GGA_C_WI  148 /* Wilson & Ivanov */

#pragma omp declare target

typedef struct {
  double a, b, c, d, k;
} gga_c_wi_params;

DEVICE
static const gga_c_wi_params dvc_wi0_params = {
  -0.44, 0.0032407, 7.8, 0.0073, 0.000311
};

DEVICE
static const gga_c_wi_params dvc_wi_params = {
  -0.00652, 0.0007, 0.21, 0.002, 0.001
};

DEVICE
static void 
dvc_gga_c_wi_init(xc_func_type *p)
{
  gga_c_wi_params *params;

  assert(p!=NULL && p->params == NULL);
  p->params = malloc(sizeof(gga_c_wi_params));
  params = (gga_c_wi_params *) (p->params);

  switch(p->info->number){
  case XC_GGA_C_WI0: 
    memcpy(params, &dvc_wi0_params, sizeof(gga_c_wi_params));
    break;
  case XC_GGA_C_WI:
    memcpy(params, &dvc_wi_params, sizeof(gga_c_wi_params));
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in gga_c_wi\n");
    exit(1);
    #endif
    break;
  }
}

#include "maple2c/gga_exc/gga_c_wi.c"
#include "work_gga_new.cu"

extern DEVICE
const xc_func_info_type dvc_xc_func_info_gga_c_wi0 = {
  XC_GGA_C_WI0,
  XC_CORRELATION,
  "Wilson & Ivanov initial version",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_Wilson1998_523, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-14,
  0, NULL, NULL,
  dvc_gga_c_wi_init, NULL,
  NULL, dvc_work_gga, NULL
};

extern DEVICE
const xc_func_info_type dvc_xc_func_info_gga_c_wi = {
  XC_GGA_C_WI,
  XC_CORRELATION,
  "Wilson & Ivanov",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_Wilson1998_523, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-10,
  0, NULL, NULL,
  dvc_gga_c_wi_init, NULL,
  NULL, dvc_work_gga, NULL
};

#pragma omp end declare target
