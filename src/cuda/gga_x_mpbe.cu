/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include "util.h"
#include "dvc_util.h"

#define XC_GGA_X_MPBE         122 /* Adamo & Barone modification to PBE             */

#pragma omp declare target

typedef struct{
  double a;
  double c1, c2, c3;
} gga_x_mpbe_params;


DEVICE
static void 
dvc_gga_x_mpbe_init(xc_func_type *p)
{
  gga_x_mpbe_params *params;

  assert(p!=NULL && p->params == NULL);
  p->params = malloc(sizeof(gga_x_mpbe_params));
  params = (gga_x_mpbe_params *) (p->params);
 
  switch(p->info->number){
  case XC_GGA_X_MPBE:
    params->a  =  0.157;
    params->c1 =  0.21951;
    params->c2 = -0.015;
    params->c3 =  0.0;
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in gga_x_mpbe\n");
    exit(1);
    #endif
    break;
  }
}


#include "maple2c/gga_exc/gga_x_mpbe.c"
#include "work_gga_new.cu"

extern DEVICE
const xc_func_info_type dvc_xc_func_info_gga_x_mpbe = {
  XC_GGA_X_MPBE,
  XC_EXCHANGE,
  "Adamo & Barone modification to PBE",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_Adamo2002_5933, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-21,
  0, NULL, NULL,
  dvc_gga_x_mpbe_init, NULL,
  NULL, dvc_work_gga, NULL
};

#pragma omp end declare target
