/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include "util.h"
#include "dvc_util.h"

#define XC_GGA_X_HJS_PBE     525 /* HJS screened exchange PBE version */
#define XC_GGA_X_HJS_PBE_SOL 526 /* HJS screened exchange PBE_SOL version */
#define XC_GGA_X_HJS_B88     527 /* HJS screened exchange B88 version */
#define XC_GGA_X_HJS_B97X    528 /* HJS screened exchange B97x version */

#pragma omp declare target

typedef struct{
  double omega;

  const double *a, *b; /* pointers to the a and b parameters */
} gga_x_hjs_params;

DEVICE
static const double dvc_a_PBE[] = 
  {0.0159941, 0.0852995, -0.160368, 0.152645, -0.0971263, 0.0422061};
DEVICE
static const double dvc_b_PBE[] = 
  {5.33319, -12.4780, 11.0988, -5.11013, 1.71468, -0.610380, 0.307555, -0.0770547, 0.0334840};

DEVICE
static const double dvc_a_PBE_sol[] = 
  {0.0047333, 0.0403304, -0.0574615, 0.0435395, -0.0216251, 0.0063721};
DEVICE
static const double dvc_b_PBE_sol[] = 
  {8.52056, -13.9885, 9.28583, -3.27287, 0.843499, -0.235543, 0.0847074, -0.0171561, 0.0050552};

DEVICE
static const double dvc_a_B88[] =
  {0.00968615, -0.0242498, 0.0259009, -0.0136606, 0.00309606, -7.32583e-5};
DEVICE
static const double dvc_b_B88[] =
  {-2.50356, 2.79656, -1.79401, 0.714888, -0.165924, 0.0118379, 0.0037806, -1.57905e-4, 1.45323e-6};

DEVICE
static const double dvc_a_B97x[] =
  {0.0027355, 0.0432970, -0.0669379, 0.0699060, -0.0474635, 0.0153092};
DEVICE
static const double dvc_b_B97x[] =
  {15.8279, -26.8145, 17.8127, -5.98246, 1.25408, -0.270783, 0.0919536, -0.0140960, 0.0045466};

DEVICE
static const double dvc_a_B88_V2[] =
  {0.0253933, -0.0673075, 0.0891476, -0.0454168, -0.00765813, 0.0142506};
DEVICE
static const double dvc_b_B88_V2[] =
  {-2.6506, 3.91108, -3.31509, 1.54485, -0.198386, -0.136112, 0.0647862, 0.0159586, -0.000245066};

DEVICE
static void
dvc_gga_x_hjs_init(xc_func_type *p)
{
  gga_x_hjs_params *params;
  
  assert(p->params == NULL);
  p->params = malloc(sizeof(gga_x_hjs_params));
  params = (gga_x_hjs_params *) (p->params);
  
  /* omega = 0.11 is set by ext_params */
  switch(p->info->number){
  case XC_GGA_X_HJS_PBE:
    params->a = dvc_a_PBE;
    params->b = dvc_b_PBE;
    break;
  case XC_GGA_X_HJS_PBE_SOL:
    params->a = dvc_a_PBE_sol;
    params->b = dvc_b_PBE_sol;
    break;
  case XC_GGA_X_HJS_B88:
    params->a = dvc_a_B88;
    params->b = dvc_b_B88;
    break;
  case XC_GGA_X_HJS_B97X:
    params->a = dvc_a_B97x;
    params->b = dvc_b_B97x;
    break;
  case XC_GGA_X_HJS_B88_V2:
    params->a = dvc_a_B88_V2;
    params->b = dvc_b_B88_V2;
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in gga_x_hjs_init\n");
    exit(1);
    #endif
    break;
  }
}

DEVICE
static func_params_type dvc_ext_params[] = {
  {"_omega", 0.11, "Screening parameter for HF"},
};

DEVICE
static void 
dvc_set_ext_params(xc_func_type *p, const double *ext_params)
{
  gga_x_hjs_params *params;

  assert(p != NULL && p->params != NULL);
  params = (gga_x_hjs_params *) (p->params);

  params->omega = dvc_get_ext_param(p->info->ext_params, ext_params, 0);
}


#include "maple2c/gga_exc/gga_x_hjs.c"
#include "work_gga_new.cu"

DEVICE
const xc_func_info_type dvc_xc_func_info_gga_x_hjs_pbe = {
  XC_GGA_X_HJS_PBE,
  XC_EXCHANGE,
  "HJS screened exchange PBE version",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_Henderson2008_194105, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  5e-12,
  1, dvc_ext_params, dvc_set_ext_params,
  dvc_gga_x_hjs_init, NULL, 
  NULL, dvc_work_gga, NULL
};

DEVICE
const xc_func_info_type dvc_xc_func_info_gga_x_hjs_pbe_sol = {
  XC_GGA_X_HJS_PBE_SOL,
  XC_EXCHANGE,
  "HJS screened exchange PBE_SOL version",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_Henderson2008_194105, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  5e-12,
  1, dvc_ext_params, dvc_set_ext_params,
  dvc_gga_x_hjs_init, NULL, 
  NULL, dvc_work_gga, NULL
};

DEVICE
const xc_func_info_type dvc_xc_func_info_gga_x_hjs_b88 = {
  XC_GGA_X_HJS_B88,
  XC_EXCHANGE,
  "HJS screened exchange B88 version",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_Henderson2008_194105, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-7, /* densities smaller than 1e-7 yield NaNs */
  1, dvc_ext_params, dvc_set_ext_params,
  dvc_gga_x_hjs_init, NULL, 
  NULL,  dvc_work_gga, NULL
};

DEVICE
const xc_func_info_type dvc_xc_func_info_gga_x_hjs_b97x = {
  XC_GGA_X_HJS_B97X,
  XC_EXCHANGE,
  "HJS screened exchange B97x version",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_Henderson2008_194105, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-10,
  1, dvc_ext_params, dvc_set_ext_params,
  dvc_gga_x_hjs_init, NULL, 
  NULL, dvc_work_gga, NULL
};

#pragma omp end declare target
