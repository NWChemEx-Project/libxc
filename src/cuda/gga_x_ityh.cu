#include "hip/hip_runtime.h"
/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include "util.h"
#include "dvc_util.h"

#define XC_GGA_X_ITYH 529 /* short-range recipe B88 functionals - erf */

#pragma omp declare target

DEVICE
static const func_params_type dvc_ext_params[] = {
  {"_omega", 0.2, "Screening parameter"},
};

DEVICE
static void 
dvc_set_ext_params(xc_func_type *p, const double *ext_params)
{
  p->cam_omega = dvc_get_ext_param(p->info->ext_params, ext_params, 0);
}

#include "maple2c/gga_exc/gga_x_ityh.c"
#include "work_gga_new.cu"

extern DEVICE
const xc_func_info_type dvc_xc_func_info_gga_x_ityh = {
  XC_GGA_X_ITYH,
  XC_EXCHANGE,
  "Short-range recipe for B88 functional - erf",
  XC_FAMILY_GGA,
  {&dvc_xc_ref_Iikura2001_3540, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-8,
  1, dvc_ext_params, dvc_set_ext_params,
  NULL, NULL, 
  NULL, dvc_work_gga, NULL
};

#pragma omp end declare target
