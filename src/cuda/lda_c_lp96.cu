/*
 Copyright (C) 2017 Susi Lehtola

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include "util.h"
#include "dvc_util.h"

#define XC_LDA_C_LP96      289   /* Liu-Parr correlation */
#define XC_LDA_K_LP96      580   /* Liu-Parr kinetic */

#pragma omp declare target

typedef struct {
  double C1, C2, C3;
} lda_c_lp96_params;

DEVICE static lda_c_lp96_params dvc_c_lp96 = {-0.0603,   0.0175, -0.00053};
DEVICE static lda_c_lp96_params dvc_k_lp96 = { 0.03777, -0.01002, 0.00039};

DEVICE static void 
dvc_lda_c_lp96_init(xc_func_type *p)
{
  lda_c_lp96_params *params;

  assert(p!=NULL && p->params == NULL);
  p->params = malloc(sizeof(lda_c_lp96_params));
  params = (lda_c_lp96_params *) (p->params);

  switch(p->info->number){
  case XC_LDA_C_LP96:
    memcpy(params, &dvc_c_lp96, sizeof(lda_c_lp96_params));
    break;
  case XC_LDA_K_LP96:
    memcpy(params, &dvc_k_lp96, sizeof(lda_c_lp96_params));
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in lda_c_lp96\n");
    exit(1);
    #endif
    break;
  }
}

#include "maple2c/lda_exc/lda_c_lp96.c"
#include "work_lda_new.cu"

DEVICE const xc_func_info_type dvc_xc_func_info_lda_c_lp96 = {
  XC_LDA_C_LP96,
  XC_CORRELATION,
  "Liu-Parr correlation",
  XC_FAMILY_LDA,
  {&dvc_xc_ref_Liu1996_2211, &dvc_xc_ref_Liu2000_29, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-16,
  0, NULL, NULL,
  dvc_lda_c_lp96_init, NULL,
  dvc_work_lda, NULL, NULL
};

DEVICE const xc_func_info_type dvc_xc_func_info_lda_k_lp96 = {
  XC_LDA_K_LP96,
  XC_KINETIC,
  "Liu-Parr kinetic",
  XC_FAMILY_LDA,
  {&dvc_xc_ref_Liu1996_2211, &dvc_xc_ref_Liu2000_29, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-16,
  0, NULL, NULL,
  dvc_lda_c_lp96_init, NULL,
  dvc_work_lda, NULL, NULL
};
#pragma omp end declare target
