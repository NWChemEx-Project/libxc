/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"
#include "dvc_util.h"

#define XC_MGGA_X_BR89_EXPLICIT    586 /* Becke-Roussel 89 with an explicit inversion of x(y), gamma = 0.8 */
#define XC_MGGA_X_BR89_EXPLICIT_1  602 /* Becke-Roussel 89 with an explicit inversion of x(y), gamma = 1.0 */

#pragma omp declare target

typedef struct{
  double gamma;
} mgga_x_br89_params;

DEVICE static const mgga_x_br89_params dvc_par_one = {1.0};

DEVICE static void
dvc_mgga_x_br89_init(xc_func_type *p)
{
  mgga_x_br89_params *params;

  assert(p != NULL && p->params == NULL);
  p->params = malloc(sizeof(mgga_x_br89_params));
  params = (mgga_x_br89_params *)p->params;

  switch(p->info->number){
  case XC_MGGA_X_BR89_EXPLICIT:
    /* default set by set_ext_params */
    break;
  case XC_MGGA_X_BR89_EXPLICIT_1:
    memcpy(params, &dvc_par_one, sizeof(mgga_x_br89_params));
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in mgga_x_br89_explicit\n");
    exit(1);
    #endif
    break;
  }
}

DEVICE static const func_params_type dvc_ext_params[] = {
  {"_gamma", 0.8, "gamma"},
};

DEVICE static void
dvc_set_ext_params(xc_func_type *p, const double *ext_params)
{
  mgga_x_br89_params *params;

  assert(p != NULL && p->params != NULL);
  params = (mgga_x_br89_params *) (p->params);

  params->gamma = dvc_get_ext_param(p->info->ext_params, ext_params, 0);
}

#include "maple2c/mgga_exc/mgga_x_br89_explicit.c"
#include "work_mgga_new.cu"

extern DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_br89_explicit = {
  XC_MGGA_X_BR89_EXPLICIT,
  XC_EXCHANGE,
  "Becke-Roussel 89 with an explicit inversion of x(y), gamma = 0.8",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Becke1989_3761, &dvc_xc_ref_Proynov2008_103, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_NEEDS_LAPLACIAN | XC_FLAGS_I_HAVE_ALL,
  1.0e-12,
  1, dvc_ext_params, dvc_set_ext_params,
  dvc_mgga_x_br89_init, NULL,
  NULL, NULL, dvc_work_mgga
};

extern DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_br89_explicit_1 = {
  XC_MGGA_X_BR89_EXPLICIT_1,
  XC_EXCHANGE,
  "Becke-Roussel 89 with an explicit inversion of x(y), gamma = 1.0",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Becke1989_3761, &dvc_xc_ref_Proynov2008_103, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_NEEDS_LAPLACIAN | XC_FLAGS_I_HAVE_ALL,
  1.0e-12,
  0, NULL, NULL,
  dvc_mgga_x_br89_init, NULL,
  NULL, NULL, dvc_work_mgga
};

#pragma omp end declare target
