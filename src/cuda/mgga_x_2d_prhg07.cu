#include "hip/hip_runtime.h"
/*
 Copyright (C) 2006-2009 J.I.J. Ojajarvi

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"
#include "dvc_util.h"

#define XC_MGGA_X_2D_PRHG07         210   /* Pittalis, Rasanen, Helbig, Gross Exchange Functional */
#define XC_MGGA_X_2D_PRHG07_PRP10   211   /* PRGH07 with PRP10 correction */

#pragma omp declare target

typedef struct xc_mgga_work_x_t {
  int   order; /* to which order should I return the derivatives */
  double rs, zeta, x, t, u;

  double f;                                   /* enhancement factor       */
  double dfdrs, dfdx, dfdt, dfdu;             /* first derivatives of f  */
  double d2fdrs2, d2fdx2, d2fdt2, d2fdu2;     /* second derivatives of zk */
  double d2fdrsx, d2fdrst, d2fdrsu, d2fdxt, d2fdxu, d2fdtu;
} xc_mgga_work_x_t;

#ifdef __HIPCC__
// In CUDA 'long double' is treated as 'double'.
// However functions on 'long double' are not permitted,
// so we define those functions here to get around that limitation.
// Because of the limitations of the CUDA language we implement
// the relevant functions using 'double' (IF CUDA ever were to
// support 'long double' this code would need to be removed to
// take advantage of such capability).
//
DEVICE inline long double expl(long double x) noexcept
{
  long double result;
  double input = x;
  result = exp(input);
  return result;
}
DEVICE inline long double logl(long double x) noexcept
{
  long double result;
  double input = x;
  result = log(input);
  return result;
}
DEVICE inline long double fabsl(long double x) noexcept
{
  if (x < 0.0) {
    return -x;
  }
  else {
    return x;
  }
}
#endif

/* Standard Newton's method */
DEVICE static double
dvc_prhg_newt(double c, double tol, double * res, int *ierr)
{
  int count;
  long double y, f, yf;
  long double ey, fp, step;
  static int max_iter = 50;

   *ierr = 1;
   if(c < -1.0)
     return 0.0;
     
   count = 0;
   
   /** We need to calculate y in different ways in different regions
   because of numerical problems. (y-1)*exp(y) is very nasty at high y
   and log(y-1)+y is very nasty at low y. **/
   if (c < 4.0) {
     y = 2.0;
     do {
       ey = expl(y);
       yf = (y-1.0)*ey;
       f = yf - c;
       fp = ey*y;
       
       step = f/fp;
       
       y -= fabsl(step) < 1.0 ? step : (step)/fabsl(step);
       y  = fabsl(y);
       
       count ++;
       *res = fabsl(f);
     } while((*res > tol) && (count < max_iter));
   }
   else {
     y = 6.0;
     c = logl(c);
     do {
       yf = logl(y-1.0)+y;
       f = yf - c;
       fp = 1.0 + 1.0/(-1.0 + y);
       
       step = f/fp;
       
       y -= fabsl(step) < 1.0 ? step : (step)/fabsl(step);
       y  = fabsl(y);
       
       count ++;
       *res = fabsl(f);
     } while((*res > tol) && (count < max_iter));
   }
   
   if(count == max_iter) *ierr=0;
   
   return y;
}

DEVICE double dvc_xc_mgga_x_2d_prhg_get_y(double C)
{
  double rhs, res, y, tol;
  int ierr;

  tol = 5e-12;

  rhs = C/M_PI;

  y = dvc_prhg_newt(rhs, tol, &res, &ierr);
  #ifndef __HIPCC__
  if(ierr == 0){
    fprintf(stderr, 
	    "Warning: Convergence not reached in PRHG functional\n"
	    "For c = %e (residual = %e)\n", C, res);
  }
  #endif

  return y;
}

DEVICE static void 
func(const xc_func_type *p, xc_mgga_work_x_t *r)
{
  double y;
  double v_PRHG, C;

  assert(p != NULL);
  
  C = 0.25*(r->u - 4.0*r->t + 0.5*r->x*r->x);
  
  y = dvc_xc_mgga_x_2d_prhg_get_y(C);
  
  v_PRHG = M_PI*dvc_xc_bessel_I0(y/2.0);
  v_PRHG /= X_FACTOR_2D_C;

  if (p->info->number == XC_MGGA_X_2D_PRHG07) {
    r->dfdrs = v_PRHG*(1.0 / 3.0); // This factor is here in order to get the correct potential through work_mgga_x.c
    r->f = v_PRHG / 2.0;
  }else if (p->info->number == XC_MGGA_X_2D_PRHG07_PRP10) {
    r->dfdrs = (v_PRHG - ((2.0*M_SQRT2)/(3.0*M_PI))*sqrt(max(2.0*r->t - 0.25*r->x*r->x, 0.0))/X_FACTOR_2D_C)*(1.0 / 3.0);
    r->f = r->dfdrs * (3.0 / 2.0);
  }

  r->dfdrs /= -r->rs; /* due to the definition of dfdrs */
  
  return;
}
#define XC_DIMENSIONS 2
#include "work_mgga_x.cu"

DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_2d_prhg07 = {
  XC_MGGA_X_2D_PRHG07,
  XC_EXCHANGE,
  "Pittalis-Rasanen-Helbig-Gross 2007",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Pittalis2007_235314, NULL, NULL, NULL, NULL},
  XC_FLAGS_2D | XC_FLAGS_NEEDS_LAPLACIAN | XC_FLAGS_HAVE_EXC | XC_FLAGS_I_HAVE_VXC,
  1.0e-23,
  0, NULL, NULL,
  NULL, NULL, 
  NULL, NULL,
  dvc_work_mgga_x,
};

DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_2d_prhg07_prp10 = {
  XC_MGGA_X_2D_PRHG07_PRP10,
  XC_EXCHANGE,
  "PRHG07 with Pittalis-Rasanen-Proetto 2010 correction",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Pittalis2007_235314, &dvc_xc_ref_Pittalis2010_115108, NULL, NULL, NULL},
  XC_FLAGS_2D | XC_FLAGS_NEEDS_LAPLACIAN | XC_FLAGS_HAVE_EXC | XC_FLAGS_I_HAVE_VXC,
  1.0e-23,
  0, NULL, NULL,
  NULL,
  NULL,
  NULL, NULL,
  dvc_work_mgga_x,
};

#pragma omp end declare target
