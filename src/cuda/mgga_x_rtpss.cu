/*
 Copyright (C) 2006-2007 M.A.L. Marques
 Copyright (C) 2018 Susi Lehtola

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"
#include "dvc_util.h"

#define XC_MGGA_X_RTPSS          299 /* Revised TPSS exchange by Garza, Bell and Head-Gordon */

#pragma omp declare target

typedef struct {
  double b, c, e, kappa, mu;
} mgga_x_rtpss_params;

DEVICE static void 
dvc_mgga_x_rtpss_init(xc_func_type *p)
{
  mgga_x_rtpss_params *params;

  assert(p!=NULL && p->params == NULL);
  p->params = malloc(sizeof(mgga_x_rtpss_params));
  params = (mgga_x_rtpss_params *)p->params;

  switch(p->info->number){
  case XC_MGGA_X_RTPSS:
    /* default set by set_ext_params */
    break;
  default:
    #ifndef __HIPCC__
    fprintf(stderr, "Internal error in mgga_x_rtpss\n");
    exit(1);
    #endif
    break;
  }
}

DEVICE static const func_params_type dvc_ext_params[] = {
  {"_b",      0.40,    "b"},
  {"_c",      1.59096, "c"},
  {"_e",      1.537,   "e"},
  {"_kappa",  0.8040,  "Asymptotic value of the enhancement function"},
  {"_mu",     0.21951, "Coefficient of the 2nd order expansion"},
};

DEVICE static void 
dvc_set_ext_params(xc_func_type *p, const double *ext_params)
{
  mgga_x_rtpss_params *params;

  assert(p != NULL && p->params != NULL);
  params = (mgga_x_rtpss_params *) (p->params);

  params->b      = dvc_get_ext_param(p->info->ext_params, ext_params, 0);
  params->c      = dvc_get_ext_param(p->info->ext_params, ext_params, 1);
  params->e      = dvc_get_ext_param(p->info->ext_params, ext_params, 2);
  params->kappa  = dvc_get_ext_param(p->info->ext_params, ext_params, 3);
  params->mu     = dvc_get_ext_param(p->info->ext_params, ext_params, 4);
}

#include "maple2c/mgga_exc/mgga_x_rtpss.c"
#include "work_mgga_new.cu"

DEVICE const xc_func_info_type dvc_xc_func_info_mgga_x_rtpss = {
  XC_MGGA_X_RTPSS,
  XC_EXCHANGE,
  "TPSS for surface adsorption",
  XC_FAMILY_MGGA,
  {&dvc_xc_ref_Garza2018_3083, NULL, NULL, NULL, NULL},
  XC_FLAGS_3D | XC_FLAGS_I_HAVE_ALL,
  1e-23,
  5, dvc_ext_params, dvc_set_ext_params,
  dvc_mgga_x_rtpss_init, NULL, 
  NULL, NULL, dvc_work_mgga,
};

#pragma omp end declare target
