/* Functions associated with those that appear in functionals.c but
 * these functions are all associated with GPU device implementations.
 */
#include "xc.h"
#include "xc_device.h"
#include "xc_extern.h"

xc_func_type *xc_func_data_device;


/* Initialize the functional parameters on the device
 */
__host__ void xc_func_init_device(xc_func_type *p)
{
    int number = xc_number_of_functionals();
    checkCuda(hipMalloc(&xc_func_data_device,number*sizeof(xc_func_type)));
    checkCuda(hipMemcpy(xc_func_data_device,p,number*sizeof(xc_func_type),hipMemcpyHostToDevice));
}

/* Free all the functional params on the device
 */
__host__ void xc_func_end_device()
{
    checkCuda(hipFree(xc_func_data_device));
}
