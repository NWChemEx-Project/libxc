/* Functions associated with those that appear in functionals.c but
 * these functions are all associated with GPU device implementations.
 */
#include "xc.h"
#include "xc_device.h"
#include "xc_extern.h"
#include "util.h"

xc_func_type *xc_func_data_device;
xc_func_info_type *xc_func_info_data_device;


/* Initialize the functional parameters on the device
 */
__host__ void xc_func_init_device(xc_func_type *p)
{
    /* An issue is that xc_func_type->info->flags is used during the functional 
       evaluation. Because the pointers xc_func_type->info are initialized on the
       host with host data they point to nothing useful on the device. This would
       cause memory access errors on the device.

       So, we need to create a copy xc_func_type data where we point ->info to
       memory locations that make sense on the device. We also need to set
       xc_func_info_type->flags to a sensible value in the table for the device.
       Then we need to copy both the xc_func_type and the xc_func_info_type tables
       to the device. 

       Once the data is safely on the device we can clean the host copies of the
       tables up.
     */
    xc_func_type      *xc_func_data_host;
    xc_func_info_type *xc_func_info_data_host;
    int number = xc_number_of_functionals();

    xc_func_data_host      = (xc_func_type *)     malloc(number*sizeof(xc_func_type));
    xc_func_info_data_host = (xc_func_info_type *)malloc(number*sizeof(xc_func_info_type));

    for (int ii=0; ii < number; ii++) xc_func_info_data_host[ii].flags = XC_FLAGS_HAVE_ALL;
    for (int ii=0; ii < number; ii++) xc_func_data_host[ii]            = p[ii];

    checkCuda(__FILE__,__LINE__,hipMalloc(&xc_func_data_device,
                                           number*sizeof(xc_func_type)));
    checkCuda(__FILE__,__LINE__,hipMalloc(&xc_func_info_data_device,
                                           number*sizeof(xc_func_info_type)));

    for (int ii=0; ii < number; ii++) {
      xc_func_data_host[ii].info = xc_func_info_data_device+ii;
    }

    checkCuda(__FILE__,__LINE__,hipMemcpy(xc_func_data_device,xc_func_data_host,
                                           number*sizeof(xc_func_type),hipMemcpyHostToDevice));
    checkCuda(__FILE__,__LINE__,hipMemcpy(xc_func_info_data_device,xc_func_info_data_host,
                                           number*sizeof(xc_func_info_type),hipMemcpyHostToDevice));

    free(xc_func_data_host);
    free(xc_func_info_data_host);
    xc_mix_func_init_cublas();
}

/* Free all the functional params on the device
 */
__host__ void xc_func_end_device()
{
    xc_mix_func_end_cublas();
    checkCuda(__FILE__,__LINE__,hipFree(xc_func_data_device));
    checkCuda(__FILE__,__LINE__,hipFree(xc_func_info_data_device));
}
