/* Functions associated with those that appear in functionals.c but
 * these functions are all associated with GPU device implementations.
 */
#include "xc_device.h"
#include "xc_func_data.cuh"

__host__ void xc_copy_func_type_to_device(xc_func_type *dest, xc_func_type *source)
{
    checkCuda(hipMemcpy(&(dest->nspin),&(source->nspin),sizeof(source->nspin), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(&(dest->cam_omega),&(source->cam_omega),sizeof(source->cam_omega), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(&(dest->cam_alpha),&(source->cam_alpha),sizeof(source->cam_alpha), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(&(dest->cam_beta),&(source->cam_beta),sizeof(source->cam_beta), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(&(dest->nlc_b),&(source->nlc_b),sizeof(source->nlc_b), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(&(dest->nlc_C),&(source->nlc_C),sizeof(source->nlc_C), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(&(dest->dim),&(source->dim),sizeof(source->dim), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(&(dest->dens_threshold),&(source->dens_threshold),sizeof(source->dens_threshold), hipMemcpyHostToDevice));
}

__host__ void xc_copy_func_params_to_device(xc_func_type *dest, xc_func_type *source, size_t params_size)
{
    checkCuda(hipMalloc((void**)&(dest->params),params_size));
    checkCuda(hipMemcpy(dest->params,source->params,params_size, hipMemcpyHostToDevice));
}

__host__ void xc_free_func_params(xc_func_type *dest)
{
    checkCuda(hipFree(dest->params));
}

/* Initialize the functional parameters on the device
 */
__host__ void xc_func_init_device(xc_func_type *p)
{
    xc_copy_func_type_to_device(&xc_func_data[p->func_rank],p);
    if (p->params != NULL)
        xc_copy_func_params_to_device(&xc_func_data[p->func_rank],p,p->sizeof_params);
    for (int ii=0; ii<p->n_func_aux; ii++)
        xc_func_init_device(p->func_aux[ii]);
}

/* Free all the functional params on the device
 */
__host__ void xc_func_end_device(xc_func_type *p)
{
    if (p->params != NULL) xc_free_func_params(&xc_func_data[p->func_rank]);
    for (int ii=0; ii<p->n_func_aux; ii++)
        xc_func_end_device(p->func_aux[ii]);
}
