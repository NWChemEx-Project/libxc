/* Functions associated with those that appear in functionals.c but
 * these functions are all associated with GPU device implementations.
 */
#include "xc.h"
#include "xc_device.h"
#include "xc_extern.h"
#include "util.h"

xc_func_type *xc_func_data_device;


/* Initialize the functional parameters on the device
 */
__host__ void xc_func_init_device(xc_func_type *p)
{
    int number = xc_number_of_functionals();
    checkCuda(__FILE__,__LINE__,hipMalloc(&xc_func_data_device,number*sizeof(xc_func_type)));
    checkCuda(__FILE__,__LINE__,hipMemcpy(xc_func_data_device,p,number*sizeof(xc_func_type),hipMemcpyHostToDevice));
    xc_mix_func_init_cublas();
}

/* Free all the functional params on the device
 */
__host__ void xc_func_end_device()
{
    xc_mix_func_end_cublas();
    checkCuda(__FILE__,__LINE__,hipFree(xc_func_data_device));
}
