#include "hip/hip_runtime.h"
/*
 Copyright (C) 2006-2018 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

/**
 * @file work_gga.cu
 * @brief This file is to be included in GGA functionals. As often these
 *        functionals are written as a function of rs and zeta, this
 *        routine performs the necessary conversions between this and a functional
 *        of rho.
 */
#ifdef __HIPCC__

#include <stdio.h>
extern xc_func_type *xc_func_data_device;

#define OUT_PARAMS_NO_EXC vrho, vsigma, v2rho2, v2rhosigma, v2sigma2, v3rho3, v3rho2sigma, v3rhosigma2, v3sigma3
#ifdef XC_NO_EXC
#define OUT_PARAMS OUT_PARAMS_NO_EXC
#else
#define OUT_PARAMS zk, OUT_PARAMS_NO_EXC
#endif

/**
 * @param[in,out] func_type: pointer to functional structure
 */
__global__ static void 
work_gga_device(const XC(func_type) *p,
                int dim_rho,    int dim_sigma,
                int dim_zk,     int dim_vrho,        int dim_vsigma,
                int dim_v2rho2, int dim_v2rhosigma,  int dim_v2sigma2,
                int dim_v3rho3, int dim_v3rho2sigma, int dim_v3rhosigma2, int dim_v3sigma3,
                int np,         const double *rho,   const double *sigma,
                double *zk,     double *vrho,        double *vsigma,
                double *v2rho2, double *v2rhosigma,  double *v2sigma2,
                double *v3rho3, double *v3rho2sigma, double *v3rhosigma2, double *v3sigma3)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if( tid < np ) {
        const double *rho_         = NULL;
        const double *sigma_       = NULL;
        double       *zk_          = NULL;
        double       *vrho_        = NULL;
        double       *vsigma_      = NULL;
        double       *v2rho2_      = NULL;
        double       *v2rhosigma_  = NULL;
        double       *v2sigma2_    = NULL;
        double       *v3rho3_      = NULL;
        double       *v3rho2sigma_ = NULL;
        double       *v3rhosigma2_ = NULL;
        double       *v3sigma3_    = NULL;
        rho_   = rho   + tid*dim_rho;
        sigma_ = sigma + tid*dim_sigma;
        if (zk          != NULL) zk_          = zk          + tid*dim_zk;
        if (vrho        != NULL) vrho_        = vrho        + tid*dim_vrho;
        if (vsigma      != NULL) vsigma_      = vsigma      + tid*dim_vsigma;
        if (v2rho2      != NULL) v2rho2_      = v2rho2      + tid*dim_v2rho2;
        if (v2rhosigma  != NULL) v2rhosigma_  = v2rhosigma  + tid*dim_v2rhosigma;
        if (v2sigma2    != NULL) v2sigma2_    = v2sigma2    + tid*dim_v2sigma2;
        if (v3rho3      != NULL) v3rho3_      = v3rho3      + tid*dim_v3rho3;
        if (v3rho2sigma != NULL) v3rho2sigma_ = v3rho2sigma + tid*dim_v3rho2sigma;
        if (v3rhosigma2 != NULL) v3rhosigma2_ = v3rhosigma2 + tid*dim_v3rhosigma2;
        if (v3sigma3    != NULL) v3sigma3_    = v3sigma3    + tid*dim_v3sigma3;
        work_gga(p,1,rho_,sigma_,zk_,vrho_,vsigma_,v2rho2_,v2rhosigma_,v2sigma2_,
                 v3rho3_,v3rho2sigma_,v3rhosigma2_,v3sigma3_);
    }
}

static void 
work_gga_offload(const XC(func_type) *p, int np, const double *rho, const double *sigma,
                 double *zk, double *vrho, double *vsigma,
                 double *v2rho2, double *v2rhosigma,  double *v2sigma2,
                 double *v3rho3, double *v3rho2sigma, double *v3rhosigma2, double *v3sigma3)
{
    const xc_dimensions *dim = &(p->dim);
    work_gga_device<<<std::ceil(np/1024.),1024>>>
                   (xc_func_data_device+p->func_rank,dim->rho,dim->sigma,
                    dim->zk,dim->vrho,dim->vsigma,
                    dim->v2rho2,dim->v2rhosigma,dim->v2sigma2,
                    dim->v3rho3,dim->v3rho2sigma,dim->v3rhosigma2,dim->v3sigma3,
                    np,rho,sigma,zk,vrho,vsigma,
                    v2rho2,v2rhosigma,v2sigma2,
                    v3rho3,v3rho2sigma,v3rhosigma2,v3sigma3);
    //hipError_t stat = hipGetLastError();
    //hipError_t stat = hipDeviceSynchronize();
    //if (stat != hipSuccess) {
    //    printf("Launch work_gga_device: %s\n",hipGetErrorString( stat ));
    //}
}

#endif
