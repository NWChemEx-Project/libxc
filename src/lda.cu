/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"

#ifdef __cplusplus
extern "C" {
#endif

/* get the lda functional */
void 
xc_lda_offload(const xc_func_type *func, int np, const double *rho, 
	       double *zk, double *vrho, double *v2rho2, double *v3rho3)
{
  const xc_dimensions *dim = &(func->dim);
  
  /* sanity check */
  if(zk != NULL && !(func->info->flags & XC_FLAGS_HAVE_EXC)){
    fprintf(stderr, "Functional '%s' does not provide an implementation of Exc\n",
	    func->info->name);
    exit(1);
  }

  if(vrho != NULL && !(func->info->flags & XC_FLAGS_HAVE_VXC)){
    fprintf(stderr, "Functional '%s' does not provide an implementation of vxc\n",
	    func->info->name);
    exit(1);
  }

  if(v2rho2 != NULL && !(func->info->flags & XC_FLAGS_HAVE_FXC)){
    fprintf(stderr, "Functional '%s' does not provide an implementation of fxc\n",
	    func->info->name);
    exit(1);
  }

  if(v3rho3 != NULL && !(func->info->flags & XC_FLAGS_HAVE_KXC)){
    fprintf(stderr, "Functional '%s' does not provide an implementation of kxc\n",
	    func->info->name);
    exit(1);
  }

  /* initialize output */
  if(zk != NULL)
    hipMemset(zk,     0, np*sizeof(double)*dim->zk);

  if(vrho != NULL)
    hipMemset(vrho,   0, np*sizeof(double)*dim->vrho);

  if(v2rho2 != NULL)
    hipMemset(v2rho2, 0, np*sizeof(double)*dim->v2rho2);

  if(v3rho3 != NULL)
    hipMemset(v3rho3, 0, np*sizeof(double)*dim->v3rho3);


  assert(func->info!=NULL && func->info->lda!=NULL);

  /* call the LDA offload routines */
  func->info->lda_offload(func, np, rho, zk, vrho, v2rho2, v3rho3);
}


/* specializations */
void
xc_lda_exc_offload(const xc_func_type *p, int np, const double *rho, double *zk)
{
  xc_lda_offload(p, np, rho, zk, NULL, NULL, NULL);
}

void
xc_lda_exc_vxc_offload(const xc_func_type *p, int np, const double *rho, double *zk, double *vrho)
{
  xc_lda_offload(p, np, rho, zk, vrho, NULL, NULL);
}

void
xc_lda_vxc_offload(const xc_func_type *p, int np, const double *rho, double *vrho)
{
  xc_lda_offload(p, np, rho, NULL, vrho, NULL, NULL);
}

void
xc_lda_fxc_offload(const xc_func_type *p, int np, const double *rho, double *v2rho2)
{
  xc_lda_offload(p, np, rho, NULL, NULL, v2rho2, NULL);
}

void
xc_lda_kxc_offload(const xc_func_type *p, int np, const double *rho, double *v3rho3)
{
  xc_lda_offload(p, np, rho, NULL, NULL, NULL, v3rho3);
}

#ifdef __cplusplus
}
#endif
