/*
 Copyright (C) 2006-2007 M.A.L. Marques

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/


#include "util.h"

#ifdef __cplusplus
extern "C" {
#endif

/* get the lda functional */
void 
xc_lda_offload(const xc_func_type *func, int np, const double *rho, 
	       double *zk, double *vrho, double *v2rho2, double *v3rho3, hipStream_t stream)
{
  const xc_dimensions *dim = &(func->dim);

  /* sanity check */
  if(zk != NULL && !(func->info->flags & XC_FLAGS_HAVE_EXC)){
    fprintf(stderr, "Functional '%s' does not provide an implementation of Exc\n",
	    func->info->name);
    exit(1);
  }

  if(vrho != NULL && !(func->info->flags & XC_FLAGS_HAVE_VXC)){
    fprintf(stderr, "Functional '%s' does not provide an implementation of vxc\n",
	    func->info->name);
    exit(1);
  }

  if(v2rho2 != NULL && !(func->info->flags & XC_FLAGS_HAVE_FXC)){
    fprintf(stderr, "Functional '%s' does not provide an implementation of fxc\n",
	    func->info->name);
    exit(1);
  }

  if(v3rho3 != NULL && !(func->info->flags & XC_FLAGS_HAVE_KXC)){
    fprintf(stderr, "Functional '%s' does not provide an implementation of kxc\n",
	    func->info->name);
    exit(1);
  }

  /* initialize output */
  if(zk != NULL)
    checkCuda(__FILE__,__LINE__,hipMemsetAsync(zk,     0, np*sizeof(double)*dim->zk, stream));

  if(vrho != NULL)
    checkCuda(__FILE__,__LINE__,hipMemsetAsync(vrho,   0, np*sizeof(double)*dim->vrho, stream));

  if(v2rho2 != NULL)
    checkCuda(__FILE__,__LINE__,hipMemsetAsync(v2rho2, 0, np*sizeof(double)*dim->v2rho2, stream));

  if(v3rho3 != NULL)
    checkCuda(__FILE__,__LINE__,hipMemsetAsync(v3rho3, 0, np*sizeof(double)*dim->v3rho3, stream));


  assert(func->info!=NULL && func->info->lda!=NULL);

  /* call the LDA offload routines */
  func->info->lda_offload(func, np, rho, zk, vrho, v2rho2, v3rho3, stream);
}


/* specializations */
void
xc_lda_exc_offload(const xc_func_type *p, int np, const double *rho, double *zk, hipStream_t stream)
{
  xc_lda_offload(p, np, rho, zk, NULL, NULL, NULL, stream);
}

void
xc_lda_exc_vxc_offload(const xc_func_type *p, int np, const double *rho, double *zk, double *vrho, hipStream_t stream)
{
  xc_lda_offload(p, np, rho, zk, vrho, NULL, NULL, stream);
}

void
xc_lda_vxc_offload(const xc_func_type *p, int np, const double *rho, double *vrho, hipStream_t stream)
{
  xc_lda_offload(p, np, rho, NULL, vrho, NULL, NULL, stream);
}

void
xc_lda_fxc_offload(const xc_func_type *p, int np, const double *rho, double *v2rho2, hipStream_t stream)
{
  xc_lda_offload(p, np, rho, NULL, NULL, v2rho2, NULL, stream);
}

void
xc_lda_kxc_offload(const xc_func_type *p, int np, const double *rho, double *v3rho3, hipStream_t stream)
{
  xc_lda_offload(p, np, rho, NULL, NULL, NULL, v3rho3, stream);
}

#ifdef __cplusplus
}
#endif
