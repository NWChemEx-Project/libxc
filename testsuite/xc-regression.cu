#include "hip/hip_runtime.h"
/*
 Copyright (C) 2006-2007 M.A.L. Marques
 Copyright (C) 2014 Susi Lehtola

 This Source Code Form is subject to the terms of the Mozilla Public
 License, v. 2.0. If a copy of the MPL was not distributed with this
 file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include <xc.h>

/* Buffer size (line length) for file reads */
#define BUFSIZE 1024

typedef struct {
  /* Amount of data points */
  int n;

  /* Input: density, gradient, laplacian and kinetic energy density */
  double *rho;
  double *sigma;
  double *lapl;
  double *tau;

  /* Output: energy density */
  double *zk;

  /* .. and potentials for density, gradient, laplacian and tau */
  double *vrho;
  double *vsigma;
  double *vlapl;
  double *vtau;

  /* ... and second derivatives */
  double *v2rho2;
  double *v2tau2;
  double *v2lapl2;
  double *v2rhotau;
  double *v2rholapl;
  double *v2lapltau;
  double *v2sigma2;
  double *v2rhosigma;
  double *v2sigmatau;
  double *v2sigmalapl;

  /* ... and third derivatives */
  double *v3rho3;
} values_t;

void allocate_memory(values_t *data, int nspin, int order)
{
  hipError_t cError = hipSuccess;
  printf("enter: allocate_memory\n");
  data->zk = NULL;
  data->vrho = NULL;
  data->vsigma = NULL;
  data->vlapl = NULL;
  data->vtau = NULL;
  data->v2rho2 = NULL;
  data->v2tau2 = NULL;
  data->v2lapl2 = NULL;
  data->v2rhotau = NULL;
  data->v2rholapl = NULL;
  data->v2lapltau = NULL;
  data->v2sigma2 = NULL;
  data->v2rhosigma = NULL;
  data->v2sigmatau = NULL;
  data->v2sigmalapl = NULL;
  data->v3rho3 = NULL;

  switch(nspin) {
    case (XC_UNPOLARIZED):
      hipMallocManaged(&data->rho,data->n*sizeof(double));
      hipMallocManaged(&data->sigma,data->n*sizeof(double));
      hipMallocManaged(&data->lapl,data->n*sizeof(double));
      hipMallocManaged(&data->tau,data->n*sizeof(double));
      switch (order) {
        case (0):
          hipMallocManaged(&data->zk,data->n*sizeof(double));
          break;
        case (1):
          hipMallocManaged(&data->vrho,data->n*sizeof(double));
          hipMallocManaged(&data->vsigma,data->n*sizeof(double));
          hipMallocManaged(&data->vlapl,data->n*sizeof(double));
          hipMallocManaged(&data->vtau,data->n*sizeof(double));
          break;
        case (2):
          hipMallocManaged(&data->v2rho2,data->n*sizeof(double));
          hipMallocManaged(&data->v2tau2,data->n*sizeof(double));
          hipMallocManaged(&data->v2lapl2,data->n*sizeof(double));
          hipMallocManaged(&data->v2rhotau,data->n*sizeof(double));
          hipMallocManaged(&data->v2rholapl,data->n*sizeof(double));
          hipMallocManaged(&data->v2lapltau,data->n*sizeof(double));
          hipMallocManaged(&data->v2sigma2,data->n*sizeof(double));
          hipMallocManaged(&data->v2rhosigma,data->n*sizeof(double));
          hipMallocManaged(&data->v2sigmatau,data->n*sizeof(double));
          hipMallocManaged(&data->v2sigmalapl,data->n*sizeof(double));
          break;
        case (3):
          hipMallocManaged(&data->v3rho3,data->n*sizeof(double));
          break;
        default:
          fprintf(stderr, "order = %i not recognized.\n", order);
          exit(2);
      }
      if ((cError=hipGetLastError())!=hipSuccess) {
          fprintf(stderr,"allocate memory: %s\n",hipGetErrorString(cError));
      }
      break;

    case (XC_POLARIZED):
      hipMallocManaged(&data->rho,2*data->n*sizeof(double));
      hipMallocManaged(&data->sigma,3*data->n*sizeof(double));
      hipMallocManaged(&data->lapl,2*data->n*sizeof(double));
      hipMallocManaged(&data->tau,2*data->n*sizeof(double));
      switch (order) {
        case (0):
          hipMallocManaged(&data->zk,data->n*sizeof(double));
          break;
        case (1):
          hipMallocManaged(&data->vrho,2*data->n*sizeof(double));
          hipMallocManaged(&data->vsigma,3*data->n*sizeof(double));
          hipMallocManaged(&data->vlapl,2*data->n*sizeof(double));
          hipMallocManaged(&data->vtau,2*data->n*sizeof(double));
          break;
        case (2):
          hipMallocManaged(&data->v2rho2,3*data->n*sizeof(double));
          hipMallocManaged(&data->v2tau2,3*data->n*sizeof(double));
          hipMallocManaged(&data->v2lapl2,3*data->n*sizeof(double));
          hipMallocManaged(&data->v2rhotau,4*data->n*sizeof(double));
          hipMallocManaged(&data->v2rholapl,4*data->n*sizeof(double));
          hipMallocManaged(&data->v2lapltau,4*data->n*sizeof(double));
          hipMallocManaged(&data->v2sigma2,6*data->n*sizeof(double));
          hipMallocManaged(&data->v2rhosigma,6*data->n*sizeof(double));
          hipMallocManaged(&data->v2sigmatau,6*data->n*sizeof(double));
          hipMallocManaged(&data->v2sigmalapl,6*data->n*sizeof(double));
          break;
        case (3):
          hipMallocManaged(&data->v3rho3,4*data->n*sizeof(double));
          break;
        default:
          fprintf(stderr, "order = %i not recognized.\n", order);
          exit(2);
      }
      if ((cError=hipGetLastError())!=hipSuccess) {
          fprintf(stderr,"allocate memory: %s\n",hipGetErrorString(cError));
      }
      break;

    default:
      fprintf(stderr, "nspin = %i not recognized.\n", nspin);
      exit(2);
  }
  printf("leave: allocate_memory\n");

}

void free_memory(values_t val)
{
  hipError_t cError = hipSuccess;
  printf("enter: free_memory\n");
  hipFree(val.rho);
  hipFree(val.sigma);
  hipFree(val.lapl);
  hipFree(val.tau);
  hipFree(val.zk);
  hipFree(val.vrho);
  hipFree(val.vsigma);
  hipFree(val.vlapl);
  hipFree(val.vtau);
  hipFree(val.v2rho2);
  hipFree(val.v2tau2);
  hipFree(val.v2lapl2);
  hipFree(val.v2rhotau);
  hipFree(val.v2rholapl);
  hipFree(val.v2lapltau);
  hipFree(val.v2sigma2);
  hipFree(val.v2rhosigma);
  hipFree(val.v2sigmatau);
  hipFree(val.v2sigmalapl);
  hipFree(val.v3rho3);
  if ((cError=hipGetLastError())!=hipSuccess) {
      fprintf(stderr,"allocate memory: %s\n",hipGetErrorString(cError));
  }
  printf("leave: free_memory\n");
}

values_t read_data(const char *file, int nspin, int order) {
  /* Format string */
  static const char fmt[]="%lf %lf %lf %lf %lf %lf %lf %lf %lf";

  /* Data buffer */
  char buf[BUFSIZE];
  char *cp;
  /* Input data file */
  FILE *in;
  /* Loop index */
  int i;
  /* Amount of points succesfully read */
  int nsucc;
  /* Returned data */
  values_t data;

  /* Helper variables */
  double rhoa, rhob;
  double sigmaaa, sigmaab, sigmabb;
  double lapla, laplb;
  double taua, taub;

  printf("enter: read_data\n");
  /* Open file */
  in=fopen(file,"r");
  if(!in) {
    fprintf(stderr,"Error opening input file %s.\n",file);
    exit(3);
  }

  /* Read amount of data points */
  cp=fgets(buf,BUFSIZE,in);
  if(cp!=buf) {
    fprintf(stderr,"Error reading amount of data points.\n");
    exit(5);
  }
  nsucc=sscanf(buf,"%i",&data.n);
  if(nsucc!=1) {
    fprintf(stderr,"Error reading amount of input data points.\n");
    exit(4);
  }

  /* Allocate memory */
  allocate_memory(&data, nspin, order);

  for(i=0;i<data.n;i++) {
    /* Next line of input */
    cp=fgets(buf,BUFSIZE,in);
    if(cp!=buf) {
      fprintf(stderr,"Read error on line %i.\n",i+1);
      free_memory(data);
      exit(5);
    }
    /* Read data */
    nsucc=sscanf(buf, fmt, &rhoa, &rhob, &sigmaaa, &sigmaab, &sigmabb,	\
		 &lapla, &laplb, &taua, &taub);

    /* Error control */
    if(nsucc!=9) {
      fprintf(stderr,"Read error on line %i: only %i entries read.\n",i+1,nsucc);
      free_memory(data);
      exit(5);
    }

    /* Store data (if clause suboptimal here but better for code clarity) */
    if(nspin==XC_POLARIZED) {
      data.rho[2*i]=rhoa;
      data.rho[2*i+1]=rhob;
      data.sigma[3*i]=sigmaaa;
      data.sigma[3*i+1]=sigmaab;
      data.sigma[3*i+2]=sigmabb;
      data.lapl[2*i]=lapla;
      data.lapl[2*i+1]=laplb;
      data.tau[2*i]=taua;
      data.tau[2*i+1]=taub;
    } else {
      /* Construct full density data from alpha and beta channels */
      data.rho[i]=rhoa + rhob;
      data.sigma[i]=sigmaaa + sigmabb + 2.0*sigmaab;
      data.lapl[i]=lapla + laplb;
      data.tau[i]=taua + taub;
    }
  }

  /* Close input file */
  fclose(in);
  printf("leave: read_data\n");

  return data;
}

__global__ void evaluate_functional(int &flags,int &family,int func_id,values_t d,int nspin,int order, int *error)
{
  int nrho,    nsigma,      nlapl, ntau;
  int nvrho,   nv2rho2,     nv3rho3;
  int          nv2rhosigma;
  int nvsigma, nv2sigma2;
  int          nv2rholapl;
  int          nv2sigmalapl;
  int nvlapl,  nv2lapl2;
  int          nv2rhotau;
  int          nv2sigmatau;
  int          nv2lapltau;
  int nvtau,   nv2tau2;
  int np;
  /* functional entity */
  xc_func_type *func;
  /* Helpers for properties that may not have been implemented */
  double *zk,  *vrho,  *v2rho2,  *v3rho3;
  double *zzk, *vvrho, *vv2rho2, *vv3rho3;
  *error = 0;
  /* Initialize functional */
  if(dvc_xc_func_init(func, func_id, nspin)) {
    // fprintf(stderr, "Functional '%d' (%s) not found.\nPlease report a bug against functional_get_number.\n", func_id, argv[1]);
    // exit(1);
    *error = 2;
  }
  /* Figure out how many components there are of each input type */
  if (nspin == 1) {  /* spin unpolarized */
      nrho         = 1;
      nsigma       = 1;
      nlapl        = 1;
      ntau         = 1;
      nvrho        = 1;
      nvsigma      = 1;
      nvlapl       = 1;
      nvtau        = 1;
      nv2rho2      = 1;
      nv2rhosigma  = 1;
      nv2rholapl   = 1;
      nv2rhotau    = 1;
      nv2sigma2    = 1;
      nv2sigmalapl = 1;
      nv2sigmatau  = 1;
      nv2lapl2     = 1;
      nv2lapltau   = 1;
      nv2tau2      = 1;
      nv3rho3      = 1;
  }
  else { /* nspin == 2; spin polarized  */
      nrho         = 2;
      nsigma       = 3;
      nlapl        = 2;
      ntau         = 2;
      nvrho        = 2;
      nvsigma      = 3;
      nvlapl       = 2;
      nvtau        = 2;
      nv2rho2      = 3;
      nv2rhosigma  = 6;
      nv2rholapl   = 3;
      nv2rhotau    = 3;
      nv2sigma2    = 6;
      nv2sigmalapl = 3;
      nv2sigmatau  = 3;
      nv2lapl2     = 3;
      nv2lapltau   = 3;
      nv2tau2      = 3;
      nv3rho3      = 4;
  }
  /* Get flags */
  flags  = func->info->flags;
  family = func->info->family;

  /* Set helpers */
  zk     = (flags & XC_FLAGS_HAVE_EXC) ? d.zk     : NULL;
  vrho   = (flags & XC_FLAGS_HAVE_VXC) ? d.vrho   : NULL;
  v2rho2 = (flags & XC_FLAGS_HAVE_FXC) ? d.v2rho2 : NULL;
  v3rho3 = (flags & XC_FLAGS_HAVE_KXC) ? d.v3rho3 : NULL;

  /* Evaluate xc functional */
  for (int ii=0; ii<d.n; ii++)
  {
      if (zk == NULL)
          zzk = NULL;
      else 
          zzk = &(zk[ii]);
      if (vrho == NULL)
          vvrho = NULL;
      else
          vvrho = &(vrho[nvrho*ii]);
      if (v2rho2 == NULL)
          vv2rho2 = NULL;
      else
          vv2rho2 = &(v2rho2[nv2rho2*ii]);
      if (v3rho3 == NULL)
          vv3rho3 = NULL;
      else
          vv3rho3 = &(v3rho3[nv3rho3*ii]);
      np = 1;
      switch(family) {
      case XC_FAMILY_LDA:
        dvc_xc_lda(func, np, &d.rho[nrho*ii], zzk, vvrho, vv2rho2, vv3rho3);
        break;
      case XC_FAMILY_GGA:
      case XC_FAMILY_HYB_GGA:
        dvc_xc_gga
               (func, np, &d.rho[nrho*ii], &d.sigma[nsigma*ii], zzk, 
               vvrho, &d.vsigma[nvsigma*ii],
               vv2rho2, &d.v2rhosigma[nv2rhosigma*ii], &d.v2sigma2[nv2sigma2*ii],
               NULL, NULL, NULL, NULL);
        break;
      case XC_FAMILY_MGGA:
      case XC_FAMILY_HYB_MGGA:
        dvc_xc_mgga
               (func, np, &d.rho[nrho*ii], &d.sigma[nsigma*ii], &d.lapl[nlapl*ii], &d.tau[ntau*ii],
                zzk, vvrho, &d.vsigma[nvsigma*ii], &d.vlapl[nvlapl*ii], &d.vtau[nvtau*ii],
                vv2rho2, &d.v2rhosigma[nv2rhosigma*ii], &d.v2rholapl[nv2rholapl*ii],
                &d.v2rhotau[nv2rhotau*ii], &d.v2sigma2[nv2sigma2*ii], &d.v2sigmalapl[nv2sigmalapl*ii],
                &d.v2sigmatau[nv2sigmatau*ii], &d.v2lapl2[nv2lapl2*ii], &d.v2lapltau[nv2lapltau*ii],
                &d.v2tau2[nv2tau2*ii],
                NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL,
                NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL);
        break;
      default:
        // fprintf(stderr,"Support for family %i not implemented.\n",family);
        // free_memory(d);
        // exit(1);
        *error = 1;
      }
  }
  dvc_xc_func_end(func);
}


/*----------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int func_id, nspin, order, i;
  //int ii;
  /* The number of different components dependent on spin polarization */
  //int nrho,    nsigma,      nlapl, ntau;
  //int nvrho,   nv2rho2,     nv3rho3;
  //int          nv2rhosigma;
  //int nvsigma, nv2sigma2;
  //int          nv2rholapl;
  //int          nv2sigmalapl;
  //int nvlapl,  nv2lapl2;
  //int          nv2rhotau;
  //int          nv2sigmatau;
  //int          nv2lapltau;
  //int nvtau,   nv2tau2;
  int error;
  /* Helpers for properties that may not have been implemented */
  //double *zk,  *vrho,  *v2rho2,  *v3rho3;
  //double *zzk, *vvrho, *vv2rho2, *vv3rho3;

  static const char efmt[] =" % .16e";
  static const char efmt2[]=" % .16e % .16e";
  static const char efmt3[]=" % .16e % .16e % .16e";
  static const char sfmt[] =" %23s";
  static const char sfmt2[]=" %23s %23s";
  static const char sfmt3[]=" %23s %23s %23s";

  if(argc != 6) {
    fprintf(stderr, "Usage:\n%s funct nspin order input output\n", argv[0]);
    exit(1);
  }

  /* Get functional id */
  func_id = xc_functional_get_number(argv[1]);
  if(func_id <= 0) {
    fprintf(stderr, "Functional '%s' not found\n", argv[1]);
    exit(1);
  }

  /* Spin-polarized or unpolarized ? */
  nspin = atoi(argv[2]);

  /* Order of derivatives to compute */
  order = atoi(argv[3]);

  /* Data array */
  values_t d;
  /* Functional evaluator */
  //xc_func_type func;
  /* Flags for functional */
  int flags;
  /* Functional family */
  int family;
  /* Output file */
  FILE *out;
  /* Output file name */
  char *fname;

  /* Read in data */
  d = read_data(argv[4], nspin, order);

  printf("call: evaluate_functional\n");
  evaluate_functional<<<1,1>>>(flags,family,func_id,d,nspin,order,&error);
  if (error) {
    fprintf(stderr,"Error in functional evaluation %d.\n",error);
    exit(1);
  }
  printf("pass: evaluate_functional\n");

  /* Open output file */
  fname = argv[5];
  out = fopen(fname,"w");
  if(!out) {
    fprintf(stderr,"Error opening output file %s.\n",fname);
    free_memory(d);
    exit(1);
  }

  /* Functional id and amount of lines in output */
  fprintf(out, "%i %i %i\n", func_id, d.n, order);

  switch (order) {
    case (0): /* energy */
      fprintf(out, sfmt, "zk");
      break;
    case (1): /* first order derivatives */
      if (nspin == XC_POLARIZED) {
        fprintf(out, sfmt2, "vrho(a)", "vrho(b)");
        if (family & (XC_FAMILY_GGA | XC_FAMILY_HYB_GGA | XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA))
          fprintf(out, sfmt3, "vsigma(aa)", "vsigma(ab)", "vsigma(bb)");
        if (family & (XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
          fprintf(out, sfmt2, "vlapl(a)", "vlapl(b)");
          fprintf(out, sfmt2, "vtau(a)", "vtau(b)");
        }
      } else {
        fprintf(out, sfmt, "vrho");
        if (family & (XC_FAMILY_GGA | XC_FAMILY_HYB_GGA | XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA))
          fprintf(out, sfmt, "vsigma");
        if(family & (XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
          fprintf(out, sfmt, "vlapl");
          fprintf(out, sfmt, "vtau");
        }
      }
      break;

    case (2): /* second order derivatives */
      if (nspin == XC_POLARIZED) {
        fprintf(out,sfmt3,"v2rho(aa)","v2rho(ab)","v2rho(bb)");
        if(family & (XC_FAMILY_GGA | XC_FAMILY_HYB_GGA | XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
          fprintf(out, sfmt3, "v2sigma2(aa-aa)", "v2sigma2(aa-ab)", "v2sigma2(aa-bb)");
          fprintf(out, sfmt3, "v2sigma2(ab-ab)", "v2sigma2(ab-bb)", "v2sigma2(bb-bb)");
          fprintf(out, sfmt3, "v2rho(a)sigma(aa)", "v2rho(a)sigma(ab)", "v2rho(a)sigma(bb)");
          fprintf(out, sfmt3, "v2rho(b)sigma(aa)", "v2rho(b)sigma(ab)", "v2rho(b)sigma(bb)");
        }
        if(family & (XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
          fprintf(out, sfmt3, "v2lapl2(aa)", "v2lapl2(ab)", "v2lapl2(bb)");
          fprintf(out, sfmt3, "v2tau2(aa)", "v2tau2(ab)", "v2tau2(bb)");
          fprintf(out, sfmt3, "v2rholapl(aa)", "v2rholapl(ab)", "v2rholapl(bb)");
          fprintf(out, sfmt3, "v2rhotau(aa)", "v2rhotau(ab)", "v2rhotau(bb)");
          fprintf(out, sfmt3, "v2lapltau(aa)", "v2lapltau(ab)", "v2lapltau(bb)");
          fprintf(out, sfmt3, "v2sigma(aa)tau(a)", "v2sigma(aa)tau(b)", "v2sigma(ab)tau(a)");
          fprintf(out, sfmt3, "v2sigma(ab)tau(b)", "v2sigma(bb)tau(a)", "v2sigma(bb)tau(b)");
          fprintf(out, sfmt3, "v2sigma(aa)lapl(a)", "v2sigma(aa)lapl(b)", "v2sigma(ab)lapl(a)");
          fprintf(out, sfmt3, "v2sigma(ab)lapl(b)", "v2sigma(bb)lapl(a)", "v2sigma(bb)lapl(b)");
        }
      } else {
        fprintf(out,sfmt,"v2rho");
        if(family & (XC_FAMILY_GGA | XC_FAMILY_HYB_GGA | XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
          fprintf(out, sfmt, "v2sigma2");
          fprintf(out, sfmt, "v2rhosigma");
        }

        if(family & (XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
          fprintf(out, sfmt, "v2lapl2");
          fprintf(out, sfmt, "v2tau2");
          fprintf(out, sfmt, "v2rholapl");
          fprintf(out, sfmt, "v2rhotau");
          fprintf(out, sfmt, "v2lapltau");
          fprintf(out, sfmt, "v2sigmatau");
          fprintf(out, sfmt, "v2sigmalapl");
        }
      }
      break;

    default: /* higher order derivatives ... to be done */
      fprintf(stderr, "order = %i not recognized.\n", order);
      exit(2);
  }
  fprintf(out,"\n");

  /* Loop over data points */
  for(i=0;i<d.n;i++) {

    switch (order) {
      case (0): /* energy */
        fprintf(out, efmt, d.zk[i]);
        break;
      case (1): /* first order derivatives */
        if (nspin == XC_POLARIZED) {
          fprintf(out, efmt2, d.vrho[2 * i], d.vrho[2 * i + 1]);
          if (family & (XC_FAMILY_GGA | XC_FAMILY_HYB_GGA | XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA))
            fprintf(out, efmt3, d.vsigma[3 * i], d.vsigma[3 * i + 1], d.vsigma[3 * i + 2]);
          if (family & (XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
            fprintf(out, efmt2, d.vlapl[2 * i], d.vlapl[2 * i + 1]);
            fprintf(out, efmt2, d.vtau[2 * i], d.vtau[2 * i + 1]);
          }
        } else {
          fprintf(out, efmt, d.vrho[i]);
          if (family & (XC_FAMILY_GGA | XC_FAMILY_HYB_GGA | XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA))
            fprintf(out, efmt, d.vsigma[i]);
          if (family & (XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
            fprintf(out, efmt, d.vlapl[i]);
            fprintf(out, efmt, d.vtau[i]);
          }
        }
        break;

      case (2): /* second order derivatives */
        if (nspin == XC_POLARIZED) {
          fprintf(out, efmt3, d.v2rho2[3*i], d.v2rho2[3*i + 1], d.v2rho2[3*i + 2]);
          if(family & (XC_FAMILY_GGA | XC_FAMILY_HYB_GGA | XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
            fprintf(out, efmt3, d.v2sigma2[6*i], d.v2sigma2[6*i + 1], d.v2sigma2[6*i + 2]);
            fprintf(out, efmt3, d.v2sigma2[6*i + 3], d.v2sigma2[6*i + 4], d.v2sigma2[6*i + 5]);
            fprintf(out, efmt3, d.v2rhosigma[6*i], d.v2rhosigma[6*i + 1], d.v2rhosigma[6*i + 2]);
            fprintf(out, efmt3, d.v2rhosigma[6*i + 3], d.v2rhosigma[6*i + 4], d.v2rhosigma[6*i + 5]);
          }
          if(family & (XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
            fprintf(out, efmt3, d.v2lapl2[3*i], d.v2lapl2[3*i + 1], d.v2lapl2[3*i + 2]);
            fprintf(out, efmt3, d.v2tau2[3*i], d.v2tau2[3*i + 1], d.v2tau2[3*i + 2]);
            fprintf(out, efmt3, d.v2rholapl[3*i], d.v2rholapl[3*i + 1], d.v2rholapl[3*i + 2]);
            fprintf(out, efmt3, d.v2rhotau[3*i], d.v2rhotau[3*i + 1], d.v2rhotau[3*i + 2]);
            fprintf(out, efmt3, d.v2lapltau[3*i], d.v2lapltau[3*i + 1], d.v2lapltau[3*i + 2]);
            fprintf(out, efmt3, d.v2sigmatau[3*i], d.v2sigmatau[3*i + 1], d.v2sigmatau[3*i + 2]);
            fprintf(out, efmt3, d.v2sigmatau[3*i + 3], d.v2sigmatau[3*i + 4], d.v2sigmatau[3*i + 5]);
            fprintf(out, efmt3, d.v2sigmalapl[3*i], d.v2sigmalapl[3*i + 1], d.v2sigmalapl[3*i + 2]);
            fprintf(out, efmt3, d.v2sigmalapl[3*i + 3], d.v2sigmalapl[3*i + 4], d.v2sigmalapl[3*i + 5]);
          }
        } else {
          fprintf(out, efmt, d.v2rho2[i]);
          if(family & (XC_FAMILY_GGA | XC_FAMILY_HYB_GGA | XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
            fprintf(out, efmt, d.v2sigma2[i]);
            fprintf(out, efmt, d.v2rhosigma[i]);
          }
          if(family & (XC_FAMILY_MGGA | XC_FAMILY_HYB_MGGA)) {
            fprintf(out, efmt, d.v2lapl2[i]);
            fprintf(out, efmt, d.v2tau2[i]);
            fprintf(out, efmt, d.v2rholapl[i]);
            fprintf(out, efmt, d.v2rhotau[i]);
            fprintf(out, efmt, d.v2lapltau[i]);
            fprintf(out, efmt, d.v2sigmatau[i]);
            fprintf(out, efmt, d.v2sigmalapl[i]);
          }
        }
        break;

     default: /* higher order derivatives ... to be done */
        fprintf(stderr, "order = %i not recognized.\n", order);
        exit(2);
    }

    fprintf(out,"\n");
  }

  free_memory(d);
  fclose(out);

  return 0;
}
